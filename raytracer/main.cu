#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""


__global__ void add(int a, int b, int* c) {
	*c = a + b;
}

void main() {
	int h_c;
	int* d_c;

	hipMalloc((void**)&d_c, sizeof(int));

	add << <1, 1 >> > (2, 7, d_c);

	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("2 + 7 = %d\n", h_c);

	hipFree(d_c);

}