#include "hip/hip_runtime.h"

#define _USE_MATH_DEFINES
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "raytracer.hpp"

#include "sampling/sampler.hpp"

#include "utility/performance_monitoring.hpp"
#include "integrators/direct_lighting_integrator.hpp"
#include "integrators/path_integrator.hpp"
#include "camera/pixel_sampler.hpp"
#include "scene/environmentmap.hpp"
#include "utility/qualifiers.hpp"
#include "utility/debugging.hpp"
#include "shapes/circle.hpp"
#include "shapes/cuboid.hpp"
#include "medium/nvdb_medium.hpp"
#include "backend/rt_backend.hpp"
#include <optix/optix_stubs.h>
#include "texture/texture_manager.hpp"

namespace rt {
  // Initializes cuRAND random number generators
  __global__ void init(CSampler* sampler, SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t samplerId = y * frame->width + x;
    sampler[samplerId].init(samplerId, 0);
  }

  __global__ void clearBuffer(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      frame->data[currentPixel + 0] = 0.f;
      frame->data[currentPixel + 1] = 0.f;
      frame->data[currentPixel + 2] = 0.f;
    }
  }

  // Raytracing
  __global__ void renderFrame(CDeviceScene* scene, CCamera* camera, CSampler* sampler, uint16_t numSamples, SDeviceFrame* frame) {
    //uint16_t y = blockIdx.y;
    //uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;


    //if (y < frame->height && x < frame->width) {
    //  //extern __shared__ char sharedScene[];
    //  //SSharedMemoryInitializer::copyScene(sharedScene, scene);

    //  uint32_t currentPixel = frame->bpp * (y * frame->width + x);
    //  uint32_t samplerId = y * frame->width + x;

    //  CPixelSampler pixelSampler(camera, x, y, &(sampler[samplerId]));
    //  //CPathIntegrator integrator((CDeviceScene*)sharedScene, &pixelSampler, &(sampler[samplerId]), numSamples);
    //  CPathIntegrator integrator(scene, &pixelSampler, &(sampler[samplerId]), numSamples);
    //  glm::vec3 L = integrator.Li();

    //  frame->data[currentPixel + 0] += L.r;
    //  frame->data[currentPixel + 1] += L.g;
    //  frame->data[currentPixel + 2] += L.b;

    //}
  }

  D_CALLABLE inline float computeTonemapFactor(SDeviceFrame* frame, uint16_t x, uint16_t y) {
    constexpr uint8_t filterSize = 11;
    float filterHalf = (float)filterSize / 2;
    float alpha = -glm::log(0.5f) / (filterHalf * filterHalf); // 0.02: From webers law
    float weights[filterSize][filterSize];
    float sum = 0.f;
    for (int8_t dX = 0; dX < filterSize; ++dX) {
      for (int8_t dY = 0; dY < filterSize; ++dY) {
        int32_t currX = x + dX - filterHalf;
        int32_t currY = y + dY - filterHalf;
        if (currX < 0 || currX >= frame->width || currY < 0 || currY >= frame->height) {
          weights[dY][dX] = 0.f;
        }
        else {
          float distance = (float)dX * dX + (float)dY * dY;
          float weight = glm::exp(-alpha * distance);
          sum += weight;
          weights[dY][dX] = weight;
        }
      }
    }

    float sigma(0.f);
    for (int8_t dX = 0; dX < filterSize; ++dX) {
      for (int8_t dY = 0; dY < filterSize; ++dY) {
        int32_t currX = x + dX - filterHalf;
        int32_t currY = y + dY - filterHalf;
        if (!(currX < 0 || currX >= frame->width || currY < 0 || currY >= frame->height)) {
          uint32_t currentPixel = frame->bpp * (currY * frame->width + currX);

          float r = frame->data[currentPixel + 0];
          float g = frame->data[currentPixel + 1];
          float b = frame->data[currentPixel + 2];
          sigma += glm::log(r + g + b) * weights[dY][dX] / sum;
        }
        
      }
    }

    return glm::exp(sigma);
  }

  __global__ void filterFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      float sigma = computeTonemapFactor(frame, x, y);
      
      frame->filtered[currentPixel + 0] = sigma;
    }
  }

  __global__ void computeGlobalTonemapping1(SDeviceFrame* frame, float* avg) {
    uint16_t y = 0;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      float divisor = frame->height * frame->width * frame->bpp;
      avg[x] = 0.f;
      for (uint16_t yIter = y; yIter < frame->height; ++yIter) {
          uint32_t currentPixel = frame->bpp * (yIter * frame->width + x);
          avg[x] += glm::log(frame->data[currentPixel + 0] + frame->data[currentPixel + 1] + frame->data[currentPixel + 2] + FLT_MIN) / divisor;
      }
    }
  }

  __global__ void computeGlobalTonemapping2(SDeviceFrame* frame, float* avg, float* tonemappingFactor) {
    float result = 0.f;
    for (uint16_t i = 0; i < frame->width; ++i) {
      result += avg[i];
    }
    *tonemappingFactor = glm::exp(result) - (frame->width * frame->height * FLT_MIN);
  }

  // Map colors to [0.0f, 1.0f]
  __global__ void applyTonemapping(SDeviceFrame* frame, float* tonemapFactor) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
    
      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      float sigma = frame->filtered[currentPixel + 0];

      frame->data[currentPixel + 0] = r / (r + *tonemapFactor);
      frame->data[currentPixel + 1] = g / (g + *tonemapFactor);
      frame->data[currentPixel + 2] = b / (b + *tonemapFactor);
    }
  }

  

  // Corrects nonliniar monitor output
  __global__ void correctGamma(SDeviceFrame* frame, float gamma) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      frame->data[currentPixel + 0] = glm::pow(r, 1 / gamma);
      frame->data[currentPixel + 1] = glm::pow(g, 1 / gamma);
      frame->data[currentPixel + 2] = glm::pow(b, 1 / gamma);
    }
  }

  // Maps [0.0f, 1.0f] to [0, 255], required for jpg/png output
  __global__ void fillByteFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      frame->dataBytes[currentPixel + 0] = glm::round(frame->data[currentPixel + 0] * 255.0f);
      frame->dataBytes[currentPixel + 1] = glm::round(frame->data[currentPixel + 1] * 255.0f);
      frame->dataBytes[currentPixel + 2] = glm::round(frame->data[currentPixel + 2] * 255.0f);
    }
  }

  Raytracer::Raytracer(uint16_t frameWidth, uint16_t frameHeight) :
    m_frameWidth(frameWidth),
    m_frameHeight(frameHeight),
    m_bpp(3),
    m_scene(),
    //m_hostCamera(frameWidth, frameHeight, 90, glm::vec3(-0.5f, 0.5f, 0.5f), glm::vec3(0.0f, 0.5f, 0.0f), glm::vec3(0.0f, 1.0f, 0.0f)),
    m_hostCamera(frameWidth, frameHeight, 90, glm::vec3(-450.f, 73.f, 450.f), glm::vec3(-10.f, 73.f, -43.f), glm::vec3(0.0f, 1.0f, 0.0f)),
    //m_hostCamera(frameWidth, frameHeight, 90, glm::vec3(-5.f, 1.f, 5.f), glm::vec3(0.f, 1.f, 0.f), glm::vec3(0.0f, 1.0f, 0.0f)),
    //m_hostCamera(frameWidth, frameHeight, 160, glm::vec3(0.10f, 0.15f, 0.01f), glm::vec3(0.0f, 0.1f, 0.0f), glm::vec3(0.0f, 1.0f, 0.0f)),
    m_numSamples(32), // higher -> less noise
    m_tonemappingFactor(100.f),
    m_gamma(2.0f),
    m_deviceCamera(nullptr),
    m_deviceFrameData(nullptr),
    m_deviceSampler(nullptr),
    m_blockSize(128) {
    // Add scene objects
    m_scene.addSceneobject(CHostSceneobject(new CCircle(glm::vec3(0.0f, 0.0f, 0.0f), FLT_MAX, glm::vec3(0.0f, 1.0f, 0.0f)), glm::vec3(0.3f, 0.3f, 0.3f), 0.99f, glm::vec3(0.1f), 0.99f, 0.99f, 1.00029f, 1.2f));
    float lightness = 0.75f / 255.0f;
    //m_scene.addSceneobject(CHostSceneobject(new CCuboid(glm::vec3(-400.f, 50.f, 300.f), glm::vec3(20.f), glm::vec3(0.f, 1.f, 0.f)), glm::vec3(lightness, lightness, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // as normal reference
    //m_scene.addSceneobject(CHostSceneobject(new Sphere(glm::vec3(2.f, 0.5f, 0.f), 0.25f, glm::vec3(0.f, 1.f, 0.f)), glm::vec3(lightness, lightness, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f));
    //m_scene.addSceneobject(CHostSceneobject(new Sphere(glm::vec3(0.f, 0.5f, 2.f), 0.25f, glm::vec3(0.f, 1.f, 0.f)), glm::vec3(lightness, lightness, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f));
    //m_scene.addSceneobject(CHostSceneobject(new Sphere(glm::vec3(-2.f, 0.5f, 0.f), 0.25f, glm::vec3(0.f, 1.f, 0.f)), glm::vec3(lightness, lightness, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f));
    //m_scene.addSceneobject(CHostSceneobject(new Sphere(glm::vec3(0.f, 0.5f, -2.f), 0.25f, glm::vec3(0.f, 1.f, 0.f)), glm::vec3(lightness, lightness, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f));

    m_scene.addSceneobject(CHostSceneobject(new CNVDBMedium("../../raytracer/assets/wdas_cloud/wdas_cloud_sixteenth.nvdb", glm::vec3(0.f, 0.f, 0.f), glm::vec3(50.f, 50.f, 50.f), 1.f, 0.0001f, glm::vec3(0.f, 200.f, 0.f), glm::vec3(0.f, 1.f, 0.f), glm::vec3(1.f)))); // volume SGGX
    //m_scene.addSceneobject(CHostSceneobject(new CNVDBMedium("../../raytracer/assets/Smoke_Plume_01/Smoke_Plume_01/embergen_smoke_plume_a_0.nvdb", glm::vec3(0.f, 0.f, 0.f), glm::vec3(50.f, 50.f, 50.f), 1.f, 0.0001f, glm::vec3(0.f, 200.f, 0.f), glm::vec3(0.f, 0.f, 1.f), glm::vec3(0.5f)))); // volume SGGX
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/teapot.obj");
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/chestnut", "AL05a.obj", glm::vec3(100.f, 0.f, 0.f), glm::vec3(0.f, 0.f, 1.f), glm::vec3(1.f));
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/chestnut", "AL05m.obj", glm::vec3(100.f, 0.f, 0.f), glm::vec3(0.f, 0.f, 1.f), glm::vec3(1.f));
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/chestnut", "AL05y.obj", glm::vec3(100.f, 0.f, 0.f), glm::vec3(0.f, 0.f, 1.f), glm::vec3(1.f));
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/pine", "scrubPine.obj", glm::vec3(-100.f, 0.f, 0.f), glm::vec3(0.f, 1.f, 0.f), glm::vec3(0.3f));
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/test", "alpha_test.obj", glm::vec3(0.f), glm::vec3(0.f, 1.f, 0.f), glm::vec3(1.f));
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/hairball", "hairball.obj", glm::vec3(0.f), glm::vec3(0.f, 1.f, 0.f), glm::vec3(1.f));
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/sponza", "sponza.obj", glm::vec3(0.f), glm::vec3(0.f, 1.f, 0.f), glm::vec3(0.1f));
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/San_Miguel", "san-miguel.obj", glm::vec3(0.f), glm::vec3(0.f, 1.f, 0.f), glm::vec3(1.f));
    //m_scene.addSceneobjectsFromAssimp("../../raytracer/assets/test", "normal_map_test.obj", glm::vec3(0.f), glm::vec3(0.f, 1.f, 0.f), glm::vec3(1.f));

    // Add environment map
    //m_scene.setEnvironmentMap(CEnvironmentMap("./../../raytracer/assets/sunflowers_1k_edit.hdr"));
    m_scene.setEnvironmentMap(CEnvironmentMap("./../../raytracer/assets/envmap.hdr"));

    allocateDeviceMemory();
    initOptix();
    copyToDevice();
    initDeviceData();
  }

  Raytracer::~Raytracer() {
    freeDeviceMemory();
    CRTBackend::instance()->release();
  }

  // Renderpipeline
  SFrame Raytracer::renderFrame(const std::function<bool()>& keyCallback) {
    //CDeviceScene* scene = m_scene.deviceScene();
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    rt::clearBuffer << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());
    bool abortRendering = false;
    for (uint16_t sample = 0; sample < m_numSamples; ++sample) {
      std::cout << "Sample " << sample + 1 << "/" << m_numSamples << std::endl;
      //CPerformanceMonitoring::startMeasurement("renderFrame");
      //rt::renderFrame << <grid, m_blockSize, sizeof(CDeviceScene) + m_scene.sceneobjects().size() * (sizeof(CDeviceSceneobject) + dataSize) >> > (scene, m_deviceCamera, m_deviceSampler, m_numSamples, m_deviceFrame);
      OPTIX_ASSERT(optixLaunch(
        CRTBackend::instance()->pipeline(),
        0,             // stream
        reinterpret_cast<hipDeviceptr_t>(m_deviceLaunchParams),
        sizeof(SLaunchParams),
        &CRTBackend::instance()->sbt(),
        m_frameWidth,  // launch width
        m_frameHeight, // launch height
        1       // launch depth
      ));
      CUDA_ASSERT(hipDeviceSynchronize());
      abortRendering = keyCallback();
      if (abortRendering) {
        return retrieveFrame();
      }
      //CPerformanceMonitoring::endMeasurement("renderFrame");
    }
    //rt::filterFrame << <grid, m_blockSize >> > (m_deviceFrame);
    //CUDA_ASSERT(hipDeviceSynchronize());

    dim3 reductionGrid(m_frameWidth / m_blockSize, 1);
    rt::computeGlobalTonemapping1 << <reductionGrid, m_blockSize >> > (m_deviceFrame, m_deviceAverage);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::computeGlobalTonemapping2 << <1, 1 >> > (m_deviceFrame, m_deviceAverage, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    ////CPerformanceMonitoring::startMeasurement("applyTonemapping");
    ////rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_tonemappingFactor);
    rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());
    ////CPerformanceMonitoring::endMeasurement("applyTonemapping");

    ////CPerformanceMonitoring::startMeasurement("correctGamma");
    rt::correctGamma << <grid, m_blockSize >> > (m_deviceFrame, m_gamma);
    CUDA_ASSERT(hipDeviceSynchronize());
    ////CPerformanceMonitoring::endMeasurement("correctGamma");

    ////CPerformanceMonitoring::startMeasurement("fillByteFrame");
    rt::fillByteFrame << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());
    //CPerformanceMonitoring::endMeasurement("fillByteFrame");

    SFrame frame = retrieveFrame();
    return frame;
  }

  SFrame Raytracer::renderPreview() {
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);

    rt::clearBuffer << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());

    OPTIX_ASSERT(optixLaunch(
      CRTBackend::instance()->pipeline(),
      0,             // stream
      reinterpret_cast<hipDeviceptr_t>(m_deviceLaunchParams),
      sizeof(SLaunchParams),
      &CRTBackend::instance()->sbt(),
      m_frameWidth,  // launch width
      m_frameHeight, // launch height
      1       // launch depth
    ));
    CUDA_ASSERT(hipDeviceSynchronize());

    dim3 reductionGrid(m_frameWidth / m_blockSize, 1);
    rt::computeGlobalTonemapping1 << <reductionGrid, m_blockSize >> > (m_deviceFrame, m_deviceAverage);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::computeGlobalTonemapping2 << <1, 1 >> > (m_deviceFrame, m_deviceAverage, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::correctGamma << <grid, m_blockSize >> > (m_deviceFrame, m_gamma);
    CUDA_ASSERT(hipDeviceSynchronize());

    SFrame frame = retrieveFrame();
    return frame;
  }

  // Distributes N spheres evenly around circle
  glm::vec3 Raytracer::getSpherePosition(float sphereRadius, uint8_t index, uint8_t maxSpheres) {
    float x = 4.0f * sphereRadius * std::cos(2 * M_PI / maxSpheres * index);
    float z = -4.0f * sphereRadius * std::sin(2 * M_PI / maxSpheres * index);
    float y = sphereRadius;
    return glm::vec3(x, y, z);
  }

  void Raytracer::updateCamera(EPressedKey pressedKeys, const glm::vec2& mouseMoveDir) {
    glm::vec3 posCamSpace(0.f);
    if (pressedKeys & EPressedKey::W) {
      posCamSpace += glm::vec3(0.f, 0.f, 0.5f);
    }
    if (pressedKeys & EPressedKey::S) {
      posCamSpace -= glm::vec3(0.f, 0.f, 0.5f);
    }
    if (pressedKeys & EPressedKey::A) {
      posCamSpace -= glm::vec3(0.5f, 0.f, 0.f);
    }
    if (pressedKeys & EPressedKey::D) {
      posCamSpace += glm::vec3(0.5f, 0.f, 0.f);
    }
    if (pressedKeys & EPressedKey::Q) {
      posCamSpace -= glm::vec3(0.f, 0.5f, 0.f);
    }
    if (pressedKeys & EPressedKey::E) {
      posCamSpace += glm::vec3(0.f, 0.5f, 0.f);
    }

    // Move camera only along along three axes around up vector
    {
      glm::vec3 viewDir = glm::vec3(m_hostCamera.viewToWorld() * glm::vec4(0.f, 0.f, -1.f, 0.f));
      glm::vec3 moveDirRight = glm::cross(viewDir, m_hostCamera.up());
      glm::vec3 moveDirForward = glm::cross(m_hostCamera.up(), moveDirRight);

      glm::mat4 moveToWorld = glm::mat4(glm::vec4(glm::normalize(moveDirRight), 0.f), glm::vec4(glm::normalize(m_hostCamera.up()), 0.f), glm::vec4(glm::normalize(moveDirForward), 0.f), glm::vec4(m_hostCamera.position(), 1.f));
      glm::vec3 posWorldSpace = glm::vec3(moveToWorld * glm::vec4(posCamSpace, 1.f));

      //glm::vec3 posWorldSpace = glm::vec3(m_hostCamera.viewToWorld() * glm::vec4(posCamSpace, 1.f));
      m_hostCamera.updatePosition(posWorldSpace);
    }

    {
      glm::vec3 viewDir(0.f, 0.f, -1.f);
      viewDir += glm::vec3(mouseMoveDir.x, mouseMoveDir.y, 0.f) * 0.03f;
      viewDir = glm::normalize(viewDir);
      glm::vec3 lookAtCamSpace = viewDir;
      glm::vec3 lookAtWorldSpace = glm::vec3(m_hostCamera.viewToWorld() * glm::vec4(lookAtCamSpace, 1.f));
      m_hostCamera.updateLookAt(lookAtWorldSpace);
    }




    hipMemcpy(m_deviceCamera, &m_hostCamera, sizeof(CCamera), hipMemcpyHostToDevice);
  }

  void Raytracer::allocateDeviceMemory() {
    m_scene.allocateDeviceMemory();
    hipMalloc(&m_deviceSampler, sizeof(CSampler) * m_frameWidth * m_frameHeight);
    hipMalloc(&m_deviceCamera, sizeof(CCamera));
    hipMalloc(&m_deviceFrame, sizeof(SDeviceFrame));
    hipMalloc(&m_deviceFrameData, sizeof(float)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp);
    hipMalloc(&m_deviceFilteredFrame, sizeof(float)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp);
    hipMalloc(&m_deviceFrameDataBytes, sizeof(uint8_t)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp);
    hipMalloc(&m_deviceAverage, sizeof(float)*m_frameWidth);
    hipMalloc(&m_deviceTonemappingValue, sizeof(float));
    hipMalloc(&m_deviceLaunchParams, sizeof(SLaunchParams));
    CTextureManager::allocateDeviceMemory();
  }

  void Raytracer::copyToDevice() {
    m_scene.copyToDevice();
    CCamera deviceCamera = m_hostCamera;
    hipMemcpy(m_deviceCamera, &deviceCamera, sizeof(CCamera), hipMemcpyHostToDevice);
    
    SDeviceFrame f;
    f.width = m_hostCamera.sensorWidth();
    f.height = m_hostCamera.sensorHeight();
    f.bpp = m_bpp;
    f.data = m_deviceFrameData;
    f.filtered = m_deviceFilteredFrame;
    f.dataBytes = m_deviceFrameDataBytes;
    hipMemcpy(m_deviceFrame, &f, sizeof(SDeviceFrame), hipMemcpyHostToDevice);

    SLaunchParams launchParams;
    launchParams.width = m_hostCamera.sensorWidth();
    launchParams.height = m_hostCamera.sensorHeight();
    launchParams.bpp = m_bpp;
    launchParams.data = m_deviceFrameData;
    launchParams.filtered = m_deviceFilteredFrame;
    launchParams.dataBytes = m_deviceFrameDataBytes;
    launchParams.scene = m_scene.deviceScene();
    launchParams.camera = m_deviceCamera;
    launchParams.sampler = m_deviceSampler;
    launchParams.numSamples = m_numSamples;
    hipMemcpy(m_deviceLaunchParams, &launchParams, sizeof(SLaunchParams), hipMemcpyHostToDevice);

    CTextureManager::copyToDevice();
  }

  void Raytracer::initDeviceData() {
    //CPerformanceMonitoring::startMeasurement("init");
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    init << <grid, m_blockSize >> > (m_deviceSampler, m_deviceFrame);
    hipError_t e = hipDeviceSynchronize();
    //CPerformanceMonitoring::endMeasurement("init");
  }

  void Raytracer::initOptix() {
    CRTBackend* rtBackend = CRTBackend::instance();
    rtBackend->init();
#ifdef DEBUG
    std::string modulePath = "cuda_to_ptx.dir/Debug/shaders.optix.ptx";
#endif
#ifdef RELEASE
    std::string modulePath = "cuda_to_ptx.dir/Release/shaders.optix.ptx";
#endif
    rtBackend->createModule(modulePath);
    rtBackend->createProgramGroups();
    rtBackend->createPipeline();
    const std::vector <SRecord<const CDeviceSceneobject*>> sbtHitRecords = m_scene.getSBTHitRecords();
    rtBackend->createSBT(sbtHitRecords);
    m_scene.buildOptixAccel();
  }

  void Raytracer::freeDeviceMemory() {
    m_scene.freeDeviceMemory();
    CUDA_ASSERT(hipFree(m_deviceCamera));
    CUDA_ASSERT(hipFree(m_deviceFrameData));
    CUDA_ASSERT(hipFree(m_deviceFrame));
    CUDA_ASSERT(hipFree(m_deviceAverage));
    CUDA_ASSERT(hipFree(m_deviceTonemappingValue));
    CUDA_ASSERT(hipFree(m_deviceLaunchParams));

    CTextureManager::freeDeviceMemory();
  }
  SFrame Raytracer::retrieveFrame() const {
    SFrame frame;
    uint32_t entries = m_frameWidth * m_frameHeight * m_bpp;
    frame.width = m_frameWidth;
    frame.height = m_frameHeight;
    frame.bpp = m_bpp;
    frame.data.resize(entries);
    CUDA_ASSERT(hipMemcpy(frame.data.data(), m_deviceFrameData, entries * sizeof(float), hipMemcpyDeviceToHost));
    frame.dataBytes.resize(entries);
    CUDA_ASSERT(hipMemcpy(frame.dataBytes.data(), m_deviceFrameDataBytes, entries * sizeof(uint8_t), hipMemcpyDeviceToHost));
    return frame;
  }
}