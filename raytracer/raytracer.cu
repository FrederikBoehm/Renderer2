#include "hip/hip_runtime.h"

#define _USE_MATH_DEFINES
#include <cmath>

#include ""

#include "raytracer.hpp"

namespace rt {
  __global__ void init(CCamera* camera) {
    camera->initCurandState();
  }

  __global__ void renderFrame(CDeviceScene* scene, CCamera* camera, SFrame* frame) {
    //SSurfaceInteraction si = scene->intersect(Ray(glm::vec3(0.0f, 1.0f, 2.0f), glm::vec3(0.0f, -1.0f, 0.0f)));
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x;
    uint16_t numSamples = 1;
    float* data = frame->data;

    if (y < frame->height && x < frame->width) {
      glm::vec3 pixelColor(0.0f);
      for (uint16_t sample = 0; sample < numSamples; ++sample) {
        SSurfaceInteraction si = scene->intersect(camera->samplePixel(x, y));
        pixelColor.r += si.surfaceAlbedo.r;
        pixelColor.g += si.surfaceAlbedo.g;
        pixelColor.b += si.surfaceAlbedo.b;
      }
      pixelColor /= numSamples;
      uint32_t currentPixel = y * frame->width + x;
      frame->data[currentPixel + 0] = pixelColor.r;
      frame->data[currentPixel + 1] = pixelColor.g;
      frame->data[currentPixel + 2] = pixelColor.b;
    }
    //uint32_t num = scene->m_numSceneobjects;
    //printf("Render Frame");
  }

  Raytracer::Raytracer() :
    m_scene(),
    m_hostCamera(1920, 1080, 90, glm::vec3(0.0f, 0.5f, 1.0f), glm::vec3(0.0f), glm::vec3(0.0f, 1.0f, 0.0f)),
    m_deviceCamera(nullptr) {
    // Add scene objects
    m_scene.addSceneobject(CHostSceneobject(EShape::PLANE, glm::vec3(0.0f, 0.0f, 0.0f), 10.0f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(1.0f)));
    float lightness = 100.0f / 255.0f;
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 0, 6), 0.05f, glm::vec3(), glm::vec3(lightness, lightness, 1.0f)));
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 1, 6), 0.05f, glm::vec3(), glm::vec3(1.0f, lightness, 1.0f)));
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 2, 6), 0.05f, glm::vec3(), glm::vec3(1.0f, lightness, lightness)));
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 3, 6), 0.05f, glm::vec3(), glm::vec3(1.0f, 1.0f, lightness)));
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 4, 6), 0.05f, glm::vec3(), glm::vec3(lightness, 1.0f, lightness)));
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 5, 6), 0.05f, glm::vec3(), glm::vec3(lightness, 1.0f, 1.0f)));

    allocateDeviceMemory();
    copyToDevice();
    initDeviceData();
  }

  Raytracer::~Raytracer() {
    freeDeviceMemory();
  }

  SHostFrame Raytracer::renderFrame() {
    // TODO: Measure execution time
    //hipDeviceSynchronize();
    //CDeviceScene* scene = m_scene.deviceScene();
    //dim3 grid(m_hostCamera.sensorWidth(), m_hostCamera.sensorHeight());
    //rt::renderFrame << <grid, 1 >> > (scene, m_deviceCamera, m_deviceFrame);
    //hipError_t error = hipDeviceSynchronize();
    SHostFrame frame = retrieveFrame();
    return frame;
  }

  glm::vec3 Raytracer::getSpherePosition(float sphereRadius, uint8_t index, uint8_t maxSpheres) {
    float x = 4.0f * sphereRadius * std::cos(2 * M_PI / maxSpheres * index);
    float z = -4.0f * sphereRadius * std::sin(2 * M_PI / maxSpheres * index);
    float y = sphereRadius;
    return glm::vec3(x, y, z);
  }

  void Raytracer::allocateDeviceMemory() {
    m_scene.allocateDeviceMemory();
    hipMalloc(&m_deviceCamera, sizeof(CCamera));
    hipMalloc(&m_deviceFrame, sizeof(SFrame));
    uint16_t bpp = 3;
    hipMalloc(&m_deviceFrameData, sizeof(float)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*bpp);
  }

  void Raytracer::copyToDevice() {
    m_scene.copyToDevice();
    hipMemcpy(m_deviceCamera, &m_hostCamera, sizeof(CCamera), hipMemcpyHostToDevice);
    
    SFrame f;
    f.width = m_hostCamera.sensorWidth();
    f.height = m_hostCamera.sensorHeight();
    f.bpp = 3; // TODO: initialize bpp for whole raytracer class
    f.data = m_deviceFrameData;
    hipMemcpy(m_deviceFrame, &f, sizeof(SFrame), hipMemcpyHostToDevice);
  }

  void Raytracer::initDeviceData() {
    init << <1, 1 >> > (m_deviceCamera);
  }

  void Raytracer::freeDeviceMemory() {
    m_scene.freeDeviceMemory();
    hipFree(m_deviceCamera);
    hipFree(m_deviceFrameData);
    hipFree(m_deviceFrame);
  }
  SHostFrame Raytracer::retrieveFrame() const {
    SHostFrame frame;
    uint32_t entries = m_hostCamera.sensorWidth() * m_hostCamera.sensorHeight() * 3;
    frame.data.resize(entries);
    hipMemcpy(frame.data.data(), m_deviceFrameData, entries * sizeof(float), hipMemcpyDeviceToHost);
    return frame;
  }
}