#include "hip/hip_runtime.h"

#define _USE_MATH_DEFINES
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "raytracer.hpp"

#include "sampling/sampler.hpp"

#include "utility/performance_monitoring.hpp"
#include "integrators/direct_lighting_integrator.hpp"
#include "integrators/path_integrator.hpp"
#include "camera/pixel_sampler.hpp"
#include "scene/environmentmap.hpp"
#include "utility/qualifiers.hpp"
#include "utility/debugging.hpp"
#include "shapes/circle.hpp"
#include "shapes/cuboid.hpp"
#include "medium/nvdb_medium.hpp"
#include "backend/rt_backend.hpp"
#include <optix/optix_stubs.h>
#include "backend/config_loader.hpp"
#include "backend/asset_manager.hpp"

namespace rt {
  // Initializes cuRAND random number generators
  __global__ void init(CSampler* sampler, SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t samplerId = y * frame->width + x;
    sampler[samplerId].init(samplerId, 0);
  }

  __global__ void clearBuffer(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      frame->data[currentPixel + 0] = 0.f;
      frame->data[currentPixel + 1] = 0.f;
      frame->data[currentPixel + 2] = 0.f;
    }
  }

  // Raytracing
  __global__ void renderFrame(CDeviceScene* scene, CCamera* camera, CSampler* sampler, uint16_t numSamples, SDeviceFrame* frame) {
    //uint16_t y = blockIdx.y;
    //uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;


    //if (y < frame->height && x < frame->width) {
    //  //extern __shared__ char sharedScene[];
    //  //SSharedMemoryInitializer::copyScene(sharedScene, scene);

    //  uint32_t currentPixel = frame->bpp * (y * frame->width + x);
    //  uint32_t samplerId = y * frame->width + x;

    //  CPixelSampler pixelSampler(camera, x, y, &(sampler[samplerId]));
    //  //CPathIntegrator integrator((CDeviceScene*)sharedScene, &pixelSampler, &(sampler[samplerId]), numSamples);
    //  CPathIntegrator integrator(scene, &pixelSampler, &(sampler[samplerId]), numSamples);
    //  glm::vec3 L = integrator.Li();

    //  frame->data[currentPixel + 0] += L.r;
    //  frame->data[currentPixel + 1] += L.g;
    //  frame->data[currentPixel + 2] += L.b;

    //}
  }

  D_CALLABLE inline float computeTonemapFactor(SDeviceFrame* frame, uint16_t x, uint16_t y) {
    constexpr uint8_t filterSize = 11;
    float filterHalf = (float)filterSize / 2;
    float alpha = -glm::log(0.5f) / (filterHalf * filterHalf); // 0.02: From webers law
    float weights[filterSize][filterSize];
    float sum = 0.f;
    for (int8_t dX = 0; dX < filterSize; ++dX) {
      for (int8_t dY = 0; dY < filterSize; ++dY) {
        int32_t currX = x + dX - filterHalf;
        int32_t currY = y + dY - filterHalf;
        if (currX < 0 || currX >= frame->width || currY < 0 || currY >= frame->height) {
          weights[dY][dX] = 0.f;
        }
        else {
          float distance = (float)dX * dX + (float)dY * dY;
          float weight = glm::exp(-alpha * distance);
          sum += weight;
          weights[dY][dX] = weight;
        }
      }
    }

    float sigma(0.f);
    for (int8_t dX = 0; dX < filterSize; ++dX) {
      for (int8_t dY = 0; dY < filterSize; ++dY) {
        int32_t currX = x + dX - filterHalf;
        int32_t currY = y + dY - filterHalf;
        if (!(currX < 0 || currX >= frame->width || currY < 0 || currY >= frame->height)) {
          uint32_t currentPixel = frame->bpp * (currY * frame->width + currX);

          float r = frame->data[currentPixel + 0];
          float g = frame->data[currentPixel + 1];
          float b = frame->data[currentPixel + 2];
          sigma += glm::log(r + g + b) * weights[dY][dX] / sum;
        }
        
      }
    }

    return glm::exp(sigma);
  }

  __global__ void filterFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      float sigma = computeTonemapFactor(frame, x, y);
      
      frame->filtered[currentPixel + 0] = sigma;
    }
  }

  __global__ void computeGlobalTonemapping1(SDeviceFrame* frame, float* avg) {
    uint16_t y = 0;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      float divisor = frame->height * frame->width * frame->bpp;
      avg[x] = 0.f;
      for (uint16_t yIter = y; yIter < frame->height; ++yIter) {
          uint32_t currentPixel = frame->bpp * (yIter * frame->width + x);
          avg[x] += glm::log(frame->data[currentPixel + 0] + frame->data[currentPixel + 1] + frame->data[currentPixel + 2] + FLT_MIN) / divisor;
      }
    }
  }

  __global__ void computeGlobalTonemapping2(SDeviceFrame* frame, float* avg, float* tonemappingFactor) {
    float result = 0.f;
    for (uint16_t i = 0; i < frame->width; ++i) {
      result += avg[i];
    }
    *tonemappingFactor = glm::exp(result) - (frame->width * frame->height * FLT_MIN);
  }

  // Map colors to [0.0f, 1.0f]
  __global__ void applyTonemapping(SDeviceFrame* frame, float* tonemapFactor) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
    
      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      float sigma = frame->filtered[currentPixel + 0];

      frame->data[currentPixel + 0] = r / (r + *tonemapFactor);
      frame->data[currentPixel + 1] = g / (g + *tonemapFactor);
      frame->data[currentPixel + 2] = b / (b + *tonemapFactor);
    }
  }

  

  // Corrects nonliniar monitor output
  __global__ void correctGamma(SDeviceFrame* frame, float gamma) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      frame->data[currentPixel + 0] = glm::pow(r, 1 / gamma);
      frame->data[currentPixel + 1] = glm::pow(g, 1 / gamma);
      frame->data[currentPixel + 2] = glm::pow(b, 1 / gamma);
    }
  }

  // Maps [0.0f, 1.0f] to [0, 255], required for jpg/png output
  __global__ void fillByteFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      frame->dataBytes[currentPixel + 0] = glm::round(frame->data[currentPixel + 0] * 255.0f);
      frame->dataBytes[currentPixel + 1] = glm::round(frame->data[currentPixel + 1] * 255.0f);
      frame->dataBytes[currentPixel + 2] = glm::round(frame->data[currentPixel + 2] * 255.0f);
    }
  }

  Raytracer::Raytracer(const SConfig& config) :
    m_frameWidth(config.frameWidth),
    m_frameHeight(config.frameHeight),
    m_bpp(config.channelsPerPixel),
    m_gamma(config.gamma),
    m_numSamples(config.samples),
    m_scene(config.scene),
    m_hostCamera(config.camera),
    m_deviceCamera(nullptr),
    m_deviceFrameData(nullptr),
    m_deviceSampler(nullptr),
    m_blockSize(128) {

    allocateDeviceMemory();
    initOptix();
    copyToDevice();
    initDeviceData();
  }

  Raytracer::~Raytracer() {
    freeDeviceMemory();
    CRTBackend::instance()->release();
    CAssetManager::release();
  }

  // Renderpipeline
  SFrame Raytracer::renderFrame(const std::function<bool()>& keyCallback) {
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    rt::clearBuffer << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());
    bool abortRendering = false;
    for (uint16_t sample = 0; sample < m_numSamples; ++sample) {
      std::cout << "Sample " << sample + 1 << "/" << m_numSamples << std::endl;
      OPTIX_ASSERT(optixLaunch(
        CRTBackend::instance()->pipeline(),
        0,             // stream
        reinterpret_cast<hipDeviceptr_t>(m_deviceLaunchParams),
        sizeof(SLaunchParams),
        &CRTBackend::instance()->sbt(),
        m_frameWidth,  // launch width
        m_frameHeight, // launch height
        1       // launch depth
      ));
      CUDA_ASSERT(hipDeviceSynchronize());
      abortRendering = keyCallback();
      if (abortRendering) {
        return retrieveFrame();
      }
    }

    dim3 reductionGrid(m_frameWidth / m_blockSize, 1);
    rt::computeGlobalTonemapping1 << <reductionGrid, m_blockSize >> > (m_deviceFrame, m_deviceAverage);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::computeGlobalTonemapping2 << <1, 1 >> > (m_deviceFrame, m_deviceAverage, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::correctGamma << <grid, m_blockSize >> > (m_deviceFrame, m_gamma);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::fillByteFrame << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());

    SFrame frame = retrieveFrame();
    return frame;
  }

  SFrame Raytracer::renderPreview() {
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);

    rt::clearBuffer << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());

    OPTIX_ASSERT(optixLaunch(
      CRTBackend::instance()->pipeline(),
      0,             // stream
      reinterpret_cast<hipDeviceptr_t>(m_deviceLaunchParams),
      sizeof(SLaunchParams),
      &CRTBackend::instance()->sbt(),
      m_frameWidth,  // launch width
      m_frameHeight, // launch height
      1       // launch depth
    ));
    CUDA_ASSERT(hipDeviceSynchronize());

    dim3 reductionGrid(m_frameWidth / m_blockSize, 1);
    rt::computeGlobalTonemapping1 << <reductionGrid, m_blockSize >> > (m_deviceFrame, m_deviceAverage);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::computeGlobalTonemapping2 << <1, 1 >> > (m_deviceFrame, m_deviceAverage, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::correctGamma << <grid, m_blockSize >> > (m_deviceFrame, m_gamma);
    CUDA_ASSERT(hipDeviceSynchronize());

    SFrame frame = retrieveFrame();
    return frame;
  }

  // Distributes N spheres evenly around circle
  glm::vec3 Raytracer::getSpherePosition(float sphereRadius, uint8_t index, uint8_t maxSpheres) {
    float x = 4.0f * sphereRadius * std::cos(2 * M_PI / maxSpheres * index);
    float z = -4.0f * sphereRadius * std::sin(2 * M_PI / maxSpheres * index);
    float y = sphereRadius;
    return glm::vec3(x, y, z);
  }

  void Raytracer::updateCamera(EPressedKey pressedKeys, const glm::vec2& mouseMoveDir) {
    glm::vec3 posCamSpace(0.f);
    if (pressedKeys & EPressedKey::W) {
      posCamSpace += glm::vec3(0.f, 0.f, 0.5f);
    }
    if (pressedKeys & EPressedKey::S) {
      posCamSpace -= glm::vec3(0.f, 0.f, 0.5f);
    }
    if (pressedKeys & EPressedKey::A) {
      posCamSpace -= glm::vec3(0.5f, 0.f, 0.f);
    }
    if (pressedKeys & EPressedKey::D) {
      posCamSpace += glm::vec3(0.5f, 0.f, 0.f);
    }
    if (pressedKeys & EPressedKey::Q) {
      posCamSpace -= glm::vec3(0.f, 0.5f, 0.f);
    }
    if (pressedKeys & EPressedKey::E) {
      posCamSpace += glm::vec3(0.f, 0.5f, 0.f);
    }

    // Move camera only along along three axes around up vector
    {
      glm::vec3 viewDir = m_hostCamera->viewToWorld() * glm::vec4(0.f, 0.f, -1.f, 0.f);
      glm::vec3 moveDirRight = glm::cross(viewDir, m_hostCamera->up());
      glm::vec3 moveDirForward = glm::cross(m_hostCamera->up(), moveDirRight);

      glm::mat4x3 moveToWorld = glm::mat4x3(glm::normalize(moveDirRight), glm::normalize(m_hostCamera->up()), glm::normalize(moveDirForward), m_hostCamera->position());
      glm::vec3 posWorldSpace = moveToWorld * glm::vec4(posCamSpace, 1.f);

      m_hostCamera->updatePosition(posWorldSpace);
    }

    {
      glm::vec3 viewDir(0.f, 0.f, -1.f);
      viewDir += glm::vec3(mouseMoveDir.x, mouseMoveDir.y, 0.f) * 0.03f;
      viewDir = glm::normalize(viewDir);
      glm::vec3 lookAtCamSpace = viewDir;
      glm::vec3 lookAtWorldSpace = m_hostCamera->viewToWorld() * glm::vec4(lookAtCamSpace, 1.f);
      m_hostCamera->updateLookAt(lookAtWorldSpace);
    }

    const glm::vec3& camPos = m_hostCamera->position();
    const glm::vec3& camLookAt = m_hostCamera->lookAt();
    printf("Camera pos: (%f, %f, %f), lookAt: (%f, %f, %f)\n", camPos.x, camPos.y, camPos.z, camLookAt.x, camLookAt.y, camLookAt.z);


    CUDA_ASSERT(hipMemcpy(m_deviceCamera, m_hostCamera.get(), sizeof(CCamera), hipMemcpyHostToDevice));
  }

  void Raytracer::allocateDeviceMemory() {
    CUDA_ASSERT(hipMalloc(&m_deviceSampler, sizeof(CSampler) * m_frameWidth * m_frameHeight));
    CUDA_ASSERT(hipMalloc(&m_deviceCamera, sizeof(CCamera)));
    CUDA_ASSERT(hipMalloc(&m_deviceFrame, sizeof(SDeviceFrame)));
    CUDA_ASSERT(hipMalloc(&m_deviceFrameData, sizeof(float)*m_hostCamera->sensorWidth()*m_hostCamera->sensorHeight()*m_bpp));
    CUDA_ASSERT(hipMalloc(&m_deviceFilteredFrame, sizeof(float)*m_hostCamera->sensorWidth()*m_hostCamera->sensorHeight()*m_bpp));
    CUDA_ASSERT(hipMalloc(&m_deviceFrameDataBytes, sizeof(uint8_t)*m_hostCamera->sensorWidth()*m_hostCamera->sensorHeight()*m_bpp));
    CUDA_ASSERT(hipMalloc(&m_deviceAverage, sizeof(float)*m_frameWidth));
    CUDA_ASSERT(hipMalloc(&m_deviceTonemappingValue, sizeof(float)));
    CUDA_ASSERT(hipMalloc(&m_deviceLaunchParams, sizeof(SLaunchParams)));
  }

  void Raytracer::copyToDevice() {
    CUDA_ASSERT(hipMemcpy(m_deviceCamera, m_hostCamera.get(), sizeof(CCamera), hipMemcpyHostToDevice));
    
    SDeviceFrame f;
    f.width = m_hostCamera->sensorWidth();
    f.height = m_hostCamera->sensorHeight();
    f.bpp = m_bpp;
    f.data = m_deviceFrameData;
    f.filtered = m_deviceFilteredFrame;
    f.dataBytes = m_deviceFrameDataBytes;
    CUDA_ASSERT(hipMemcpy(m_deviceFrame, &f, sizeof(SDeviceFrame), hipMemcpyHostToDevice));

    SLaunchParams launchParams;
    launchParams.width = m_hostCamera->sensorWidth();
    launchParams.height = m_hostCamera->sensorHeight();
    launchParams.bpp = m_bpp;
    launchParams.data = m_deviceFrameData;
    launchParams.filtered = m_deviceFilteredFrame;
    launchParams.dataBytes = m_deviceFrameDataBytes;
    launchParams.scene = m_scene->deviceScene();
    launchParams.camera = m_deviceCamera;
    launchParams.sampler = m_deviceSampler;
    launchParams.numSamples = m_numSamples;
    CUDA_ASSERT(hipMemcpy(m_deviceLaunchParams, &launchParams, sizeof(SLaunchParams), hipMemcpyHostToDevice));

  }

  void Raytracer::initDeviceData() {
    //CPerformanceMonitoring::startMeasurement("init");
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    init << <grid, m_blockSize >> > (m_deviceSampler, m_deviceFrame);
    hipError_t e = hipDeviceSynchronize();
    //CPerformanceMonitoring::endMeasurement("init");
  }

  void Raytracer::initOptix() {
    CRTBackend* rtBackend = CRTBackend::instance();
    rtBackend->init();
#ifdef DEBUG
    std::string modulePath = "cuda_to_ptx.dir/Debug/shaders.optix.ptx";
#endif
#ifdef RELEASE
    std::string modulePath = "cuda_to_ptx.dir/Release/shaders.optix.ptx";
#endif
    rtBackend->createModule(modulePath);
    rtBackend->createProgramGroups();
    rtBackend->createPipeline();
    const std::vector <SRecord<const CDeviceSceneobject*>> sbtHitRecords = m_scene->getSBTHitRecords();
    rtBackend->createSBT(sbtHitRecords);
    m_scene->buildOptixAccel();
  }

  void Raytracer::freeDeviceMemory() {
    CUDA_ASSERT(hipFree(m_deviceCamera));
    CUDA_ASSERT(hipFree(m_deviceFrameData));
    CUDA_ASSERT(hipFree(m_deviceFrame));
    CUDA_ASSERT(hipFree(m_deviceAverage));
    CUDA_ASSERT(hipFree(m_deviceTonemappingValue));
    CUDA_ASSERT(hipFree(m_deviceLaunchParams));

  }
  SFrame Raytracer::retrieveFrame() const {
    SFrame frame;
    uint32_t entries = m_frameWidth * m_frameHeight * m_bpp;
    frame.width = m_frameWidth;
    frame.height = m_frameHeight;
    frame.bpp = m_bpp;
    frame.data.resize(entries);
    CUDA_ASSERT(hipMemcpy(frame.data.data(), m_deviceFrameData, entries * sizeof(float), hipMemcpyDeviceToHost));
    frame.dataBytes.resize(entries);
    CUDA_ASSERT(hipMemcpy(frame.dataBytes.data(), m_deviceFrameDataBytes, entries * sizeof(uint8_t), hipMemcpyDeviceToHost));
    return frame;
  }
}