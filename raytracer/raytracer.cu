#include "hip/hip_runtime.h"

#define _USE_MATH_DEFINES
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "raytracer.hpp"

#include "sampling/sampler.hpp"

#include "utility/performance_monitoring.hpp"
#include "integrators/direct_lighting_integrator.hpp"
#include "integrators/path_integrator.hpp"
#include "camera/pixel_sampler.hpp"
#include "scene/environmentmap.hpp"
#include "utility/qualifiers.hpp"
#include "utility/debugging.hpp"
#include "shapes/circle.hpp"
#include "shapes/cuboid.hpp"
#include "medium/nvdb_medium.hpp"
#include "backend/rt_backend.hpp"
#include <optix/optix_stubs.h>
#include "texture/texture_manager.hpp"
#include "backend/config_loader.hpp"

namespace rt {
  // Initializes cuRAND random number generators
  __global__ void init(CSampler* sampler, SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t samplerId = y * frame->width + x;
    sampler[samplerId].init(samplerId, 0);
  }

  __global__ void clearBuffer(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      frame->data[currentPixel + 0] = 0.f;
      frame->data[currentPixel + 1] = 0.f;
      frame->data[currentPixel + 2] = 0.f;
    }
  }

  // Raytracing
  __global__ void renderFrame(CDeviceScene* scene, CCamera* camera, CSampler* sampler, uint16_t numSamples, SDeviceFrame* frame) {
    //uint16_t y = blockIdx.y;
    //uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;


    //if (y < frame->height && x < frame->width) {
    //  //extern __shared__ char sharedScene[];
    //  //SSharedMemoryInitializer::copyScene(sharedScene, scene);

    //  uint32_t currentPixel = frame->bpp * (y * frame->width + x);
    //  uint32_t samplerId = y * frame->width + x;

    //  CPixelSampler pixelSampler(camera, x, y, &(sampler[samplerId]));
    //  //CPathIntegrator integrator((CDeviceScene*)sharedScene, &pixelSampler, &(sampler[samplerId]), numSamples);
    //  CPathIntegrator integrator(scene, &pixelSampler, &(sampler[samplerId]), numSamples);
    //  glm::vec3 L = integrator.Li();

    //  frame->data[currentPixel + 0] += L.r;
    //  frame->data[currentPixel + 1] += L.g;
    //  frame->data[currentPixel + 2] += L.b;

    //}
  }

  D_CALLABLE inline float computeTonemapFactor(SDeviceFrame* frame, uint16_t x, uint16_t y) {
    constexpr uint8_t filterSize = 11;
    float filterHalf = (float)filterSize / 2;
    float alpha = -glm::log(0.5f) / (filterHalf * filterHalf); // 0.02: From webers law
    float weights[filterSize][filterSize];
    float sum = 0.f;
    for (int8_t dX = 0; dX < filterSize; ++dX) {
      for (int8_t dY = 0; dY < filterSize; ++dY) {
        int32_t currX = x + dX - filterHalf;
        int32_t currY = y + dY - filterHalf;
        if (currX < 0 || currX >= frame->width || currY < 0 || currY >= frame->height) {
          weights[dY][dX] = 0.f;
        }
        else {
          float distance = (float)dX * dX + (float)dY * dY;
          float weight = glm::exp(-alpha * distance);
          sum += weight;
          weights[dY][dX] = weight;
        }
      }
    }

    float sigma(0.f);
    for (int8_t dX = 0; dX < filterSize; ++dX) {
      for (int8_t dY = 0; dY < filterSize; ++dY) {
        int32_t currX = x + dX - filterHalf;
        int32_t currY = y + dY - filterHalf;
        if (!(currX < 0 || currX >= frame->width || currY < 0 || currY >= frame->height)) {
          uint32_t currentPixel = frame->bpp * (currY * frame->width + currX);

          float r = frame->data[currentPixel + 0];
          float g = frame->data[currentPixel + 1];
          float b = frame->data[currentPixel + 2];
          sigma += glm::log(r + g + b) * weights[dY][dX] / sum;
        }
        
      }
    }

    return glm::exp(sigma);
  }

  __global__ void filterFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      float sigma = computeTonemapFactor(frame, x, y);
      
      frame->filtered[currentPixel + 0] = sigma;
    }
  }

  __global__ void computeGlobalTonemapping1(SDeviceFrame* frame, float* avg) {
    uint16_t y = 0;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      float divisor = frame->height * frame->width * frame->bpp;
      avg[x] = 0.f;
      for (uint16_t yIter = y; yIter < frame->height; ++yIter) {
          uint32_t currentPixel = frame->bpp * (yIter * frame->width + x);
          avg[x] += glm::log(frame->data[currentPixel + 0] + frame->data[currentPixel + 1] + frame->data[currentPixel + 2] + FLT_MIN) / divisor;
      }
    }
  }

  __global__ void computeGlobalTonemapping2(SDeviceFrame* frame, float* avg, float* tonemappingFactor) {
    float result = 0.f;
    for (uint16_t i = 0; i < frame->width; ++i) {
      result += avg[i];
    }
    *tonemappingFactor = glm::exp(result) - (frame->width * frame->height * FLT_MIN);
  }

  // Map colors to [0.0f, 1.0f]
  __global__ void applyTonemapping(SDeviceFrame* frame, float* tonemapFactor) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
    
      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      float sigma = frame->filtered[currentPixel + 0];

      frame->data[currentPixel + 0] = r / (r + *tonemapFactor);
      frame->data[currentPixel + 1] = g / (g + *tonemapFactor);
      frame->data[currentPixel + 2] = b / (b + *tonemapFactor);
    }
  }

  

  // Corrects nonliniar monitor output
  __global__ void correctGamma(SDeviceFrame* frame, float gamma) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      frame->data[currentPixel + 0] = glm::pow(r, 1 / gamma);
      frame->data[currentPixel + 1] = glm::pow(g, 1 / gamma);
      frame->data[currentPixel + 2] = glm::pow(b, 1 / gamma);
    }
  }

  // Maps [0.0f, 1.0f] to [0, 255], required for jpg/png output
  __global__ void fillByteFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      frame->dataBytes[currentPixel + 0] = glm::round(frame->data[currentPixel + 0] * 255.0f);
      frame->dataBytes[currentPixel + 1] = glm::round(frame->data[currentPixel + 1] * 255.0f);
      frame->dataBytes[currentPixel + 2] = glm::round(frame->data[currentPixel + 2] * 255.0f);
    }
  }

  Raytracer::Raytracer(const char* configPath) :
    m_deviceCamera(nullptr),
    m_deviceFrameData(nullptr),
    m_deviceSampler(nullptr),
    m_blockSize(128) {

    SConfig config = CConfigLoader::loadConfig(configPath);
    m_frameWidth = config.frameWidth;
    m_frameHeight = config.frameHeight;
    m_bpp = config.channelsPerPixel;
    m_gamma = config.gamma;
    m_numSamples = config.samples;
    m_scene = std::move(config.scene);
    m_hostCamera = std::move(config.camera);

    allocateDeviceMemory();
    initOptix();
    copyToDevice();
    initDeviceData();
  }

  Raytracer::~Raytracer() {
    freeDeviceMemory();
    CRTBackend::instance()->release();
  }

  // Renderpipeline
  SFrame Raytracer::renderFrame(const std::function<bool()>& keyCallback) {
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    rt::clearBuffer << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());
    bool abortRendering = false;
    for (uint16_t sample = 0; sample < m_numSamples; ++sample) {
      std::cout << "Sample " << sample + 1 << "/" << m_numSamples << std::endl;
      OPTIX_ASSERT(optixLaunch(
        CRTBackend::instance()->pipeline(),
        0,             // stream
        reinterpret_cast<hipDeviceptr_t>(m_deviceLaunchParams),
        sizeof(SLaunchParams),
        &CRTBackend::instance()->sbt(),
        m_frameWidth,  // launch width
        m_frameHeight, // launch height
        1       // launch depth
      ));
      CUDA_ASSERT(hipDeviceSynchronize());
      abortRendering = keyCallback();
      if (abortRendering) {
        return retrieveFrame();
      }
    }

    dim3 reductionGrid(m_frameWidth / m_blockSize, 1);
    rt::computeGlobalTonemapping1 << <reductionGrid, m_blockSize >> > (m_deviceFrame, m_deviceAverage);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::computeGlobalTonemapping2 << <1, 1 >> > (m_deviceFrame, m_deviceAverage, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::correctGamma << <grid, m_blockSize >> > (m_deviceFrame, m_gamma);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::fillByteFrame << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());

    SFrame frame = retrieveFrame();
    return frame;
  }

  SFrame Raytracer::renderPreview() {
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);

    rt::clearBuffer << <grid, m_blockSize >> > (m_deviceFrame);
    CUDA_ASSERT(hipDeviceSynchronize());

    OPTIX_ASSERT(optixLaunch(
      CRTBackend::instance()->pipeline(),
      0,             // stream
      reinterpret_cast<hipDeviceptr_t>(m_deviceLaunchParams),
      sizeof(SLaunchParams),
      &CRTBackend::instance()->sbt(),
      m_frameWidth,  // launch width
      m_frameHeight, // launch height
      1       // launch depth
    ));
    CUDA_ASSERT(hipDeviceSynchronize());

    dim3 reductionGrid(m_frameWidth / m_blockSize, 1);
    rt::computeGlobalTonemapping1 << <reductionGrid, m_blockSize >> > (m_deviceFrame, m_deviceAverage);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::computeGlobalTonemapping2 << <1, 1 >> > (m_deviceFrame, m_deviceAverage, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_deviceTonemappingValue);
    CUDA_ASSERT(hipDeviceSynchronize());

    rt::correctGamma << <grid, m_blockSize >> > (m_deviceFrame, m_gamma);
    CUDA_ASSERT(hipDeviceSynchronize());

    SFrame frame = retrieveFrame();
    return frame;
  }

  // Distributes N spheres evenly around circle
  glm::vec3 Raytracer::getSpherePosition(float sphereRadius, uint8_t index, uint8_t maxSpheres) {
    float x = 4.0f * sphereRadius * std::cos(2 * M_PI / maxSpheres * index);
    float z = -4.0f * sphereRadius * std::sin(2 * M_PI / maxSpheres * index);
    float y = sphereRadius;
    return glm::vec3(x, y, z);
  }

  void Raytracer::updateCamera(EPressedKey pressedKeys, const glm::vec2& mouseMoveDir) {
    glm::vec3 posCamSpace(0.f);
    if (pressedKeys & EPressedKey::W) {
      posCamSpace += glm::vec3(0.f, 0.f, 0.5f);
    }
    if (pressedKeys & EPressedKey::S) {
      posCamSpace -= glm::vec3(0.f, 0.f, 0.5f);
    }
    if (pressedKeys & EPressedKey::A) {
      posCamSpace -= glm::vec3(0.5f, 0.f, 0.f);
    }
    if (pressedKeys & EPressedKey::D) {
      posCamSpace += glm::vec3(0.5f, 0.f, 0.f);
    }
    if (pressedKeys & EPressedKey::Q) {
      posCamSpace -= glm::vec3(0.f, 0.5f, 0.f);
    }
    if (pressedKeys & EPressedKey::E) {
      posCamSpace += glm::vec3(0.f, 0.5f, 0.f);
    }

    // Move camera only along along three axes around up vector
    {
      glm::vec3 viewDir = glm::vec3(m_hostCamera.viewToWorld() * glm::vec4(0.f, 0.f, -1.f, 0.f));
      glm::vec3 moveDirRight = glm::cross(viewDir, m_hostCamera.up());
      glm::vec3 moveDirForward = glm::cross(m_hostCamera.up(), moveDirRight);

      glm::mat4 moveToWorld = glm::mat4(glm::vec4(glm::normalize(moveDirRight), 0.f), glm::vec4(glm::normalize(m_hostCamera.up()), 0.f), glm::vec4(glm::normalize(moveDirForward), 0.f), glm::vec4(m_hostCamera.position(), 1.f));
      glm::vec3 posWorldSpace = glm::vec3(moveToWorld * glm::vec4(posCamSpace, 1.f));

      //glm::vec3 posWorldSpace = glm::vec3(m_hostCamera.viewToWorld() * glm::vec4(posCamSpace, 1.f));
      m_hostCamera.updatePosition(posWorldSpace);
    }

    {
      glm::vec3 viewDir(0.f, 0.f, -1.f);
      viewDir += glm::vec3(mouseMoveDir.x, mouseMoveDir.y, 0.f) * 0.03f;
      viewDir = glm::normalize(viewDir);
      glm::vec3 lookAtCamSpace = viewDir;
      glm::vec3 lookAtWorldSpace = glm::vec3(m_hostCamera.viewToWorld() * glm::vec4(lookAtCamSpace, 1.f));
      m_hostCamera.updateLookAt(lookAtWorldSpace);
    }




    CUDA_ASSERT(hipMemcpy(m_deviceCamera, &m_hostCamera, sizeof(CCamera), hipMemcpyHostToDevice));
  }

  void Raytracer::allocateDeviceMemory() {
    m_scene.allocateDeviceMemory();
    CUDA_ASSERT(hipMalloc(&m_deviceSampler, sizeof(CSampler) * m_frameWidth * m_frameHeight));
    CUDA_ASSERT(hipMalloc(&m_deviceCamera, sizeof(CCamera)));
    CUDA_ASSERT(hipMalloc(&m_deviceFrame, sizeof(SDeviceFrame)));
    CUDA_ASSERT(hipMalloc(&m_deviceFrameData, sizeof(float)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp));
    CUDA_ASSERT(hipMalloc(&m_deviceFilteredFrame, sizeof(float)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp));
    CUDA_ASSERT(hipMalloc(&m_deviceFrameDataBytes, sizeof(uint8_t)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp));
    CUDA_ASSERT(hipMalloc(&m_deviceAverage, sizeof(float)*m_frameWidth));
    CUDA_ASSERT(hipMalloc(&m_deviceTonemappingValue, sizeof(float)));
    CUDA_ASSERT(hipMalloc(&m_deviceLaunchParams, sizeof(SLaunchParams)));
    CTextureManager::allocateDeviceMemory();
  }

  void Raytracer::copyToDevice() {
    m_scene.copyToDevice();
    CCamera deviceCamera = m_hostCamera;
    CUDA_ASSERT(hipMemcpy(m_deviceCamera, &deviceCamera, sizeof(CCamera), hipMemcpyHostToDevice));
    
    SDeviceFrame f;
    f.width = m_hostCamera.sensorWidth();
    f.height = m_hostCamera.sensorHeight();
    f.bpp = m_bpp;
    f.data = m_deviceFrameData;
    f.filtered = m_deviceFilteredFrame;
    f.dataBytes = m_deviceFrameDataBytes;
    CUDA_ASSERT(hipMemcpy(m_deviceFrame, &f, sizeof(SDeviceFrame), hipMemcpyHostToDevice));

    SLaunchParams launchParams;
    launchParams.width = m_hostCamera.sensorWidth();
    launchParams.height = m_hostCamera.sensorHeight();
    launchParams.bpp = m_bpp;
    launchParams.data = m_deviceFrameData;
    launchParams.filtered = m_deviceFilteredFrame;
    launchParams.dataBytes = m_deviceFrameDataBytes;
    launchParams.scene = m_scene.deviceScene();
    launchParams.camera = m_deviceCamera;
    launchParams.sampler = m_deviceSampler;
    launchParams.numSamples = m_numSamples;
    CUDA_ASSERT(hipMemcpy(m_deviceLaunchParams, &launchParams, sizeof(SLaunchParams), hipMemcpyHostToDevice));

    CTextureManager::copyToDevice();
  }

  void Raytracer::initDeviceData() {
    //CPerformanceMonitoring::startMeasurement("init");
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    init << <grid, m_blockSize >> > (m_deviceSampler, m_deviceFrame);
    hipError_t e = hipDeviceSynchronize();
    //CPerformanceMonitoring::endMeasurement("init");
  }

  void Raytracer::initOptix() {
    CRTBackend* rtBackend = CRTBackend::instance();
    rtBackend->init();
#ifdef DEBUG
    std::string modulePath = "cuda_to_ptx.dir/Debug/shaders.optix.ptx";
#endif
#ifdef RELEASE
    std::string modulePath = "cuda_to_ptx.dir/Release/shaders.optix.ptx";
#endif
    rtBackend->createModule(modulePath);
    rtBackend->createProgramGroups();
    rtBackend->createPipeline();
    const std::vector <SRecord<const CDeviceSceneobject*>> sbtHitRecords = m_scene.getSBTHitRecords();
    rtBackend->createSBT(sbtHitRecords);
    m_scene.buildOptixAccel();
  }

  void Raytracer::freeDeviceMemory() {
    m_scene.freeDeviceMemory();
    CUDA_ASSERT(hipFree(m_deviceCamera));
    CUDA_ASSERT(hipFree(m_deviceFrameData));
    CUDA_ASSERT(hipFree(m_deviceFrame));
    CUDA_ASSERT(hipFree(m_deviceAverage));
    CUDA_ASSERT(hipFree(m_deviceTonemappingValue));
    CUDA_ASSERT(hipFree(m_deviceLaunchParams));

    CTextureManager::freeDeviceMemory();
  }
  SFrame Raytracer::retrieveFrame() const {
    SFrame frame;
    uint32_t entries = m_frameWidth * m_frameHeight * m_bpp;
    frame.width = m_frameWidth;
    frame.height = m_frameHeight;
    frame.bpp = m_bpp;
    frame.data.resize(entries);
    CUDA_ASSERT(hipMemcpy(frame.data.data(), m_deviceFrameData, entries * sizeof(float), hipMemcpyDeviceToHost));
    frame.dataBytes.resize(entries);
    CUDA_ASSERT(hipMemcpy(frame.dataBytes.data(), m_deviceFrameDataBytes, entries * sizeof(uint8_t), hipMemcpyDeviceToHost));
    return frame;
  }
}