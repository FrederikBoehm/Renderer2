#include "hip/hip_runtime.h"

#define _USE_MATH_DEFINES
#include <cmath>
#include <iostream>

#include ""

#include "raytracer.hpp"

#include "sampling/sampler.hpp"

#include "utility/performance_monitoring.hpp"

namespace rt {
  // Initializes cuRAND random number generators
  __global__ void init(CSampler* sampler, SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t samplerId = y * frame->width + x;
    sampler[samplerId].init(samplerId, 0);
  }

  // Raytracing
  __global__ void renderFrame(CDeviceScene* scene, CCamera* camera, CSampler* sampler, uint16_t numSamples, SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      uint32_t samplerId = y * frame->width + x;

      Ray eyeRay = camera->samplePixel(x, y, sampler[samplerId]);

      SSurfaceInteraction si = scene->intersect(eyeRay);
      if (si.hitInformation.hit) {
        if (si.material.Le() != glm::vec3(0.0f)) { // Hit on light source
          glm::vec3 le = si.material.Le() / (float)numSamples;
          frame->data[currentPixel + 0] += le.r;
          frame->data[currentPixel + 1] += le.g;
          frame->data[currentPixel + 2] += le.b;
        }
        else {
          glm::vec3 tangentSpaceDirection = sampler[samplerId].uniformSampleHemisphere();
          // Construct tangent space
          glm::vec3 notN = normalize(glm::vec3(si.hitInformation.normal.x + 1.0f, si.hitInformation.normal.x + 2.0f, si.hitInformation.normal.x + 3.0f));
          glm::vec3 tangent = glm::normalize(glm::cross(notN, si.hitInformation.normal));
          glm::vec3 bitangent = glm::normalize(glm::cross(si.hitInformation.normal, tangent));

          glm::mat4 tangentToWorld(glm::vec4(tangent, 0.0f), glm::vec4(si.hitInformation.normal, 0.0f), glm::vec4(bitangent, 0.0f), glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
          glm::mat4 worldToTangent = glm::inverse(tangentToWorld);
          glm::vec3 worldSpaceDirection = glm::normalize(glm::vec3(tangentToWorld * glm::vec4(tangentSpaceDirection, 0.0f)));

          Ray shadowRay = Ray(si.hitInformation.pos + FLT_EPSILON * si.hitInformation.normal, worldSpaceDirection);
          SSurfaceInteraction si2 = scene->intersect(shadowRay);

          Ray eyeRayTangent = eyeRay.transform(worldToTangent);

          glm::vec3 f = si.material.f(si.hitInformation, -eyeRayTangent.m_direction, tangentSpaceDirection);
          glm::vec3 Le = si2.material.Le();
          float cosine = glm::max(glm::dot(si.hitInformation.normal, shadowRay.m_direction), 0.0f);
          float pdf = sampler[currentPixel].uniformHemispherePdf();

          glm::vec3 L = f * Le * cosine / ((float)numSamples * pdf);
          frame->data[currentPixel + 0] += L.r;
          frame->data[currentPixel + 1] += L.g;
          frame->data[currentPixel + 2] += L.b;
        }
      }
    }
  }

  // Map colors to [0.0f, 1.0f]
  __global__ void applyTonemapping(SDeviceFrame* frame, float tonemapFactor) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
    
      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      frame->data[currentPixel + 0] = r / (r + tonemapFactor);
      frame->data[currentPixel + 1] = g / (g + tonemapFactor);
      frame->data[currentPixel + 2] = b / (b + tonemapFactor);
    }
  }

  // Corrects nonliniar monitor output
  __global__ void correctGamma(SDeviceFrame* frame, float gamma) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      frame->data[currentPixel + 0] = glm::pow(r, 1 / gamma);
      frame->data[currentPixel + 1] = glm::pow(g, 1 / gamma);
      frame->data[currentPixel + 2] = glm::pow(b, 1 / gamma);
    }
  }

  // Maps [0.0f, 1.0f] to [0, 255], required for jpg/png output
  __global__ void fillByteFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      frame->dataBytes[currentPixel + 0] = glm::round(frame->data[currentPixel + 0] * 255.0f);
      frame->dataBytes[currentPixel + 1] = glm::round(frame->data[currentPixel + 1] * 255.0f);
      frame->dataBytes[currentPixel + 2] = glm::round(frame->data[currentPixel + 2] * 255.0f);
    }
  }

  Raytracer::Raytracer(uint16_t frameWidth, uint16_t frameHeight) :
    m_frameWidth(frameWidth),
    m_frameHeight(frameHeight),
    m_bpp(3),
    m_scene(),
    m_hostCamera(frameWidth, frameHeight, 90, glm::vec3(0.0f, 0.25f, 0.5f), glm::vec3(0.0f), glm::vec3(0.0f, 1.0f, 0.0f)),
    m_numSamples(100), // higher -> less noise
    m_tonemappingFactor(1.0f),
    m_gamma(2.0f),
    m_deviceCamera(nullptr),
    m_deviceFrameData(nullptr),
    m_deviceSampler(nullptr),
    m_blockSize(128) {
    // Add scene objects
    m_scene.addSceneobject(CHostSceneobject(EShape::PLANE, glm::vec3(0.0f, 0.0f, 0.0f), 5000.f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(0.7f), 0.99f, glm::vec3(0.8f), 0.99f, 0.99f, 1.00029f, 1.2f));
    float lightness = 50.0f / 255.0f;
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 0, 6), 0.05f, glm::vec3(), glm::vec3(lightness, lightness, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // blue sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 1, 6), 0.05f, glm::vec3(), glm::vec3(0.85f, lightness, 0.85f), 0.01f,  glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // violet sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 2, 6), 0.05f, glm::vec3(), glm::vec3(0.85f, lightness, lightness), 0.01f,  glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // red sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 3, 6), 0.05f, glm::vec3(), glm::vec3(0.85f, 0.85f, lightness), 0.01f,  glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // yellow sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 4, 6), 0.05f, glm::vec3(), glm::vec3(lightness, 0.85f, lightness), 0.01f,  glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // green sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.05f, 5, 6), 0.05f, glm::vec3(), glm::vec3(lightness, 0.85f, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // cyan sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::PLANE, glm::vec3(0.0f, 0.3f, 0.0f), 0.3f, glm::vec3(0.0f, -1.0f, 0.0f), glm::vec3(3.0f))); // Light

    allocateDeviceMemory();
    copyToDevice();
    initDeviceData();
  }

  Raytracer::~Raytracer() {
    freeDeviceMemory();
  }

  // Renderpipeline
  SFrame Raytracer::renderFrame() {
    CDeviceScene* scene = m_scene.deviceScene();
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    for (uint16_t sample = 0; sample < m_numSamples; ++sample) {
      std::cout << "Sample " << sample + 1 << "/" << m_numSamples << std::endl;
      //CPerformanceMonitoring::startMeasurement("renderFrame");
      rt::renderFrame << <grid, m_blockSize >> > (scene, m_deviceCamera, m_deviceSampler, m_numSamples, m_deviceFrame);
      hipDeviceSynchronize();
      //CPerformanceMonitoring::endMeasurement("renderFrame");
    }
    //CPerformanceMonitoring::startMeasurement("applyTonemapping");
    rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_tonemappingFactor);
    hipDeviceSynchronize();
    //CPerformanceMonitoring::endMeasurement("applyTonemapping");

    //CPerformanceMonitoring::startMeasurement("correctGamma");
    rt::correctGamma << <grid, m_blockSize >> > (m_deviceFrame, m_gamma);
    hipDeviceSynchronize();
    //CPerformanceMonitoring::endMeasurement("correctGamma");

    //CPerformanceMonitoring::startMeasurement("fillByteFrame");
    rt::fillByteFrame << <grid, m_blockSize >> > (m_deviceFrame);
    hipDeviceSynchronize();
    //CPerformanceMonitoring::endMeasurement("fillByteFrame");

    SFrame frame = retrieveFrame();
    return frame;
  }

  // Distributes N spheres evenly around circle
  glm::vec3 Raytracer::getSpherePosition(float sphereRadius, uint8_t index, uint8_t maxSpheres) {
    float x = 4.0f * sphereRadius * std::cos(2 * M_PI / maxSpheres * index);
    float z = -4.0f * sphereRadius * std::sin(2 * M_PI / maxSpheres * index);
    float y = sphereRadius;
    return glm::vec3(x, y, z);
  }

  void Raytracer::allocateDeviceMemory() {
    m_scene.allocateDeviceMemory();
    hipMalloc(&m_deviceSampler, sizeof(CSampler) * m_frameWidth * m_frameHeight);
    hipMalloc(&m_deviceCamera, sizeof(CCamera));
    hipMalloc(&m_deviceFrame, sizeof(SDeviceFrame));
    hipMalloc(&m_deviceFrameData, sizeof(float)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp);
    hipMalloc(&m_deviceFrameDataBytes, sizeof(uint8_t)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp);
  }

  void Raytracer::copyToDevice() {
    m_scene.copyToDevice();
    CCamera deviceCamera = m_hostCamera;
    hipMemcpy(m_deviceCamera, &deviceCamera, sizeof(CCamera), hipMemcpyHostToDevice);
    
    SDeviceFrame f;
    f.width = m_hostCamera.sensorWidth();
    f.height = m_hostCamera.sensorHeight();
    f.bpp = m_bpp;
    f.data = m_deviceFrameData;
    f.dataBytes = m_deviceFrameDataBytes;
    hipMemcpy(m_deviceFrame, &f, sizeof(SDeviceFrame), hipMemcpyHostToDevice);
  }

  void Raytracer::initDeviceData() {
    //CPerformanceMonitoring::startMeasurement("init");
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    init << <grid, m_blockSize >> > (m_deviceSampler, m_deviceFrame);
    hipDeviceSynchronize();
    //CPerformanceMonitoring::endMeasurement("init");
  }

  void Raytracer::freeDeviceMemory() {
    m_scene.freeDeviceMemory();
    hipFree(m_deviceCamera);
    hipFree(m_deviceFrameData);
    hipFree(m_deviceFrame);
  }
  SFrame Raytracer::retrieveFrame() const {
    SFrame frame;
    uint32_t entries = m_frameWidth * m_frameHeight * m_bpp;
    frame.width = m_frameWidth;
    frame.height = m_frameHeight;
    frame.bpp = m_bpp;
    frame.data.resize(entries);
    hipMemcpy(frame.data.data(), m_deviceFrameData, entries * sizeof(float), hipMemcpyDeviceToHost);
    frame.dataBytes.resize(entries);
    hipMemcpy(frame.dataBytes.data(), m_deviceFrameDataBytes, entries * sizeof(uint8_t), hipMemcpyDeviceToHost);
    return frame;
  }
}