#include "hip/hip_runtime.h"

#define _USE_MATH_DEFINES
#include <cmath>
#include <iostream>

#include ""

#include "raytracer.hpp"

#include "sampling/sampler.hpp"

#include "utility/performance_monitoring.hpp"
#include "integrators/direct_lighting_integrator.hpp"
#include "integrators/path_integrator.hpp"
#include "camera/pixel_sampler.hpp"
#include "scene/environmentmap.hpp"
#include "utility/qualifiers.hpp"
#include "utility/debugging.hpp"

namespace rt {
  // Initializes cuRAND random number generators
  __global__ void init(CSampler* sampler, SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t samplerId = y * frame->width + x;
    sampler[samplerId].init(samplerId, 0);
  }

  // Raytracing
  __global__ void renderFrame(CDeviceScene* scene, CCamera* camera, CSampler* sampler, uint16_t numSamples, SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;


    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      uint32_t samplerId = y * frame->width + x;

      CPixelSampler pixelSampler(camera, x, y, &(sampler[samplerId]));
      CPathIntegrator integrator(scene, &pixelSampler, &(sampler[samplerId]), numSamples);
      glm::vec3 L = integrator.Li();


      frame->data[currentPixel + 0] += L.r;
      frame->data[currentPixel + 1] += L.g;
      frame->data[currentPixel + 2] += L.b;

    }
  }

  D_CALLABLE inline float computeTonemapFactor(SDeviceFrame* frame, uint16_t x, uint16_t y) {
    constexpr uint8_t filterSize = 11;
    float filterHalf = (float)filterSize / 2;
    float alpha = -glm::log(0.5f) / (filterHalf * filterHalf); // 0.02: From webers law
    float weights[filterSize][filterSize];
    float sum = 0.f;
    for (int8_t dX = 0; dX < filterSize; ++dX) {
      for (int8_t dY = 0; dY < filterSize; ++dY) {
        int32_t currX = x + dX - filterHalf;
        int32_t currY = y + dY - filterHalf;
        if (currX < 0 || currX >= frame->width || currY < 0 || currY >= frame->height) {
          weights[dY][dX] = 0.f;
        }
        else {
          float distance = (float)dX * dX + (float)dY * dY;
          float weight = glm::exp(-alpha * distance);
          sum += weight;
          weights[dY][dX] = weight;
        }
      }
    }

    float sigma(0.f);
    for (int8_t dX = 0; dX < filterSize; ++dX) {
      for (int8_t dY = 0; dY < filterSize; ++dY) {
        int32_t currX = x + dX - filterHalf;
        int32_t currY = y + dY - filterHalf;
        if (!(currX < 0 || currX >= frame->width || currY < 0 || currY >= frame->height)) {
          uint32_t currentPixel = frame->bpp * (currY * frame->width + currX);

          float r = frame->data[currentPixel + 0];
          float g = frame->data[currentPixel + 1];
          float b = frame->data[currentPixel + 2];
          sigma += glm::log(r + g + b) * weights[dY][dX] / sum;
        }
        
      }
    }

    return glm::exp(sigma);
  }

  __global__ void filterFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
      float sigma = computeTonemapFactor(frame, x, y);
      
      frame->filtered[currentPixel + 0] = sigma;
    }
  }

  __global__ void computeGlobalTonemapping1(SDeviceFrame* frame, float* avg) {
    uint16_t y = 0;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      float divisor = frame->height * frame->width * frame->bpp;
      avg[x] = 0.f;
      for (uint16_t yIter = y; yIter < frame->height; ++yIter) {
          uint32_t currentPixel = frame->bpp * (yIter * frame->width + x);
          avg[x] += glm::log(frame->data[currentPixel + 0] + frame->data[currentPixel + 1] + frame->data[currentPixel + 2] + FLT_MIN) / divisor;
      }
    }
  }

  __global__ void computeGlobalTonemapping2(SDeviceFrame* frame, float* avg, float* tonemappingFactor) {
    float result = 0.f;
    for (uint16_t i = 0; i < frame->width; ++i) {
      result += avg[i];
    }
    *tonemappingFactor = glm::exp(result) - (frame->width * frame->height * FLT_MIN);
  }

  // Map colors to [0.0f, 1.0f]
  __global__ void applyTonemapping(SDeviceFrame* frame, float* tonemapFactor) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);
    
      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      float sigma = frame->filtered[currentPixel + 0];

      frame->data[currentPixel + 0] = r / (r + *tonemapFactor);
      frame->data[currentPixel + 1] = g / (g + *tonemapFactor);
      frame->data[currentPixel + 2] = b / (b + *tonemapFactor);
    }
  }

  

  // Corrects nonliniar monitor output
  __global__ void correctGamma(SDeviceFrame* frame, float gamma) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      float r = frame->data[currentPixel + 0];
      float g = frame->data[currentPixel + 1];
      float b = frame->data[currentPixel + 2];

      frame->data[currentPixel + 0] = glm::pow(r, 1 / gamma);
      frame->data[currentPixel + 1] = glm::pow(g, 1 / gamma);
      frame->data[currentPixel + 2] = glm::pow(b, 1 / gamma);
    }
  }

  // Maps [0.0f, 1.0f] to [0, 255], required for jpg/png output
  __global__ void fillByteFrame(SDeviceFrame* frame) {
    uint16_t y = blockIdx.y;
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < frame->height && x < frame->width) {
      uint32_t currentPixel = frame->bpp * (y * frame->width + x);

      frame->dataBytes[currentPixel + 0] = glm::round(frame->data[currentPixel + 0] * 255.0f);
      frame->dataBytes[currentPixel + 1] = glm::round(frame->data[currentPixel + 1] * 255.0f);
      frame->dataBytes[currentPixel + 2] = glm::round(frame->data[currentPixel + 2] * 255.0f);
    }
  }

  Raytracer::Raytracer(uint16_t frameWidth, uint16_t frameHeight) :
    m_frameWidth(frameWidth),
    m_frameHeight(frameHeight),
    m_bpp(3),
    m_scene(),
    m_hostCamera(frameWidth, frameHeight, 90, glm::vec3(-0.5f, 0.25f, 0.5f), glm::vec3(0.0f, 0.1f, 0.0f), glm::vec3(0.0f, 1.0f, 0.0f)),
    //m_hostCamera(frameWidth, frameHeight, 160, glm::vec3(0.10f, 0.15f, 0.01f), glm::vec3(0.0f, 0.1f, 0.0f), glm::vec3(0.0f, 1.0f, 0.0f)),
    m_numSamples(300), // higher -> less noise
    m_tonemappingFactor(100.f),
    m_gamma(2.0f),
    m_deviceCamera(nullptr),
    m_deviceFrameData(nullptr),
    m_deviceSampler(nullptr),
    m_blockSize(128) {
    // Add scene objects
    m_scene.addSceneobject(CHostSceneobject(EShape::PLANE, glm::vec3(0.0f, 0.0f, 0.0f), 5000.f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(0.3f, 0.3f, 0.3f), 0.99f, glm::vec3(0.1f), 0.99f, 0.99f, 1.00029f, 1.2f));
    float lightness = 0.75f / 255.0f;
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.08f, 0, 6), 0.08f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(lightness, lightness, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // blue sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.08f, 1, 6), 0.08f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(0.85f, lightness, 0.85f), 0.01f,  glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // violet sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.08f, 2, 6), 0.08f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(0.85f, lightness, lightness), 0.01f,  glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // red sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.08f, 3, 6), 0.08f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(0.85f, 0.85f, lightness), 0.01f,  glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // yellow sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.08f, 4, 6), 0.08f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(lightness, 0.85f, lightness), 0.01f,  glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // green sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, getSpherePosition(0.08f, 5, 6), 0.08f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(lightness, 0.85f, 0.85f), 0.01f, glm::vec3(0.9f), 0.01f, 0.01f, 1.00029f, 1.5f)); // cyan sphere
    m_scene.addSceneobject(CHostSceneobject(EShape::SPHERE, glm::vec3(0.f, 0.15f, 0.0f), 0.15f, glm::vec3(0.0f, 1.0f, 0.0f), glm::vec3(0.f, 0.0f, 0.0f), glm::vec3(10.f, 10.f, 10.0f), 0.99f)); // volume
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, glm::vec3(0.0f, 0.3f, 0.0f), 0.2f, glm::vec3(0.0f, -1.0f, 0.0f), glm::vec3(1.0f))); // Light
    //glm::vec3 light1Pos = getSpherePosition(0.1f, 0, 6) + glm::vec3(0.0f, 0.2f, 0.0f);
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, light1Pos, 0.05f, -glm::normalize(light1Pos), glm::vec3(10.0f)));
    //glm::vec3 light2Pos = getSpherePosition(0.1f, 1, 6) + glm::vec3(0.0f, 0.2f, 0.0f);
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, light2Pos, 0.05f, -glm::normalize(light2Pos), glm::vec3(10.0f)));
    //glm::vec3 light3Pos = getSpherePosition(0.1f, 2, 6) + glm::vec3(0.0f, 0.2f, 0.0f);
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, light3Pos, 0.05f, -glm::normalize(light3Pos), glm::vec3(10.0f)));
    //glm::vec3 light4Pos = getSpherePosition(0.1f, 3, 6) + glm::vec3(0.0f, 0.2f, 0.0f);
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, light4Pos, 0.05f, -glm::normalize(light4Pos), glm::vec3(10.0f)));
    //glm::vec3 light5Pos = getSpherePosition(0.1f, 4, 6) + glm::vec3(0.0f, 0.2f, 0.0f);
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, light5Pos, 0.05f, -glm::normalize(light5Pos), glm::vec3(10.0f)));
    //glm::vec3 light6Pos = getSpherePosition(0.1f, 5, 6) + glm::vec3(0.0f, 0.2f, 0.0f);
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, light6Pos, 0.05f, -glm::normalize(light6Pos), glm::vec3(10.0f)));

    //glm::vec3 light2Pos = getSpherePosition(0.1f, 1, 6) + glm::vec3(0.0f, 0.1f, 0.0f);
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, light2Pos, 0.05f, -glm::normalize(light2Pos), glm::vec3(2.0f, 0.0f, 0.5f)));
    //glm::vec3 light3Pos = getSpherePosition(0.1f, 2, 6) + glm::vec3(0.0f, 0.1f, 0.0f);
    //m_scene.addLightsource(CHostSceneobject(EShape::PLANE, light3Pos, 0.05f, -glm::normalize(light3Pos), glm::vec3(2.0f)));

    // Add environment map
    //m_scene.setEnvironmentMap(CEnvironmentMap("./../../raytracer/assets/sunflowers_1k_edit.hdr"));
    m_scene.setEnvironmentMap(CEnvironmentMap("./../../raytracer/assets/envmap.hdr"));

    allocateDeviceMemory();
    copyToDevice();
    initDeviceData();
  }

  Raytracer::~Raytracer() {
    freeDeviceMemory();
  }

  // Renderpipeline
  SFrame Raytracer::renderFrame() {
    CDeviceScene* scene = m_scene.deviceScene();
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    for (uint16_t sample = 0; sample < m_numSamples; ++sample) {
      std::cout << "Sample " << sample + 1 << "/" << m_numSamples << std::endl;
      //CPerformanceMonitoring::startMeasurement("renderFrame");
      rt::renderFrame << <grid, m_blockSize >> > (scene, m_deviceCamera, m_deviceSampler, m_numSamples, m_deviceFrame);
      GPU_ASSERT(hipDeviceSynchronize());
      //CPerformanceMonitoring::endMeasurement("renderFrame");
    }
    rt::filterFrame << <grid, m_blockSize >> > (m_deviceFrame);
    GPU_ASSERT(hipDeviceSynchronize());

    dim3 reductionGrid(m_frameWidth / m_blockSize, 1);;
    rt::computeGlobalTonemapping1 << <reductionGrid, m_blockSize >> > (m_deviceFrame, m_deviceAverage);
    GPU_ASSERT(hipDeviceSynchronize());

    rt::computeGlobalTonemapping2 << <1, 1 >> > (m_deviceFrame, m_deviceAverage, m_deviceTonemappingValue);
    GPU_ASSERT(hipDeviceSynchronize());

    //CPerformanceMonitoring::startMeasurement("applyTonemapping");
    //rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_tonemappingFactor);
    rt::applyTonemapping << <grid, m_blockSize >> > (m_deviceFrame, m_deviceTonemappingValue);
    GPU_ASSERT(hipDeviceSynchronize());
    //CPerformanceMonitoring::endMeasurement("applyTonemapping");

    //CPerformanceMonitoring::startMeasurement("correctGamma");
    rt::correctGamma << <grid, m_blockSize >> > (m_deviceFrame, m_gamma);
    GPU_ASSERT(hipDeviceSynchronize());
    //CPerformanceMonitoring::endMeasurement("correctGamma");

    //CPerformanceMonitoring::startMeasurement("fillByteFrame");
    rt::fillByteFrame << <grid, m_blockSize >> > (m_deviceFrame);
    GPU_ASSERT(hipDeviceSynchronize());
    //CPerformanceMonitoring::endMeasurement("fillByteFrame");

    SFrame frame = retrieveFrame();
    return frame;
  }

  // Distributes N spheres evenly around circle
  glm::vec3 Raytracer::getSpherePosition(float sphereRadius, uint8_t index, uint8_t maxSpheres) {
    float x = 4.0f * sphereRadius * std::cos(2 * M_PI / maxSpheres * index);
    float z = -4.0f * sphereRadius * std::sin(2 * M_PI / maxSpheres * index);
    float y = sphereRadius;
    return glm::vec3(x, y, z);
  }

  void Raytracer::allocateDeviceMemory() {
    m_scene.allocateDeviceMemory();
    hipMalloc(&m_deviceSampler, sizeof(CSampler) * m_frameWidth * m_frameHeight);
    hipMalloc(&m_deviceCamera, sizeof(CCamera));
    hipMalloc(&m_deviceFrame, sizeof(SDeviceFrame));
    hipMalloc(&m_deviceFrameData, sizeof(float)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp);
    hipMalloc(&m_deviceFilteredFrame, sizeof(float)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp);
    hipMalloc(&m_deviceFrameDataBytes, sizeof(uint8_t)*m_hostCamera.sensorWidth()*m_hostCamera.sensorHeight()*m_bpp);
    hipMalloc(&m_deviceAverage, sizeof(float)*m_frameWidth);
    hipMalloc(&m_deviceTonemappingValue, sizeof(float));
  }

  void Raytracer::copyToDevice() {
    m_scene.copyToDevice();
    CCamera deviceCamera = m_hostCamera;
    hipMemcpy(m_deviceCamera, &deviceCamera, sizeof(CCamera), hipMemcpyHostToDevice);
    
    SDeviceFrame f;
    f.width = m_hostCamera.sensorWidth();
    f.height = m_hostCamera.sensorHeight();
    f.bpp = m_bpp;
    f.data = m_deviceFrameData;
    f.filtered = m_deviceFilteredFrame;
    f.dataBytes = m_deviceFrameDataBytes;
    hipMemcpy(m_deviceFrame, &f, sizeof(SDeviceFrame), hipMemcpyHostToDevice);
  }

  void Raytracer::initDeviceData() {
    //CPerformanceMonitoring::startMeasurement("init");
    dim3 grid(m_frameWidth / m_blockSize, m_frameHeight);
    init << <grid, m_blockSize >> > (m_deviceSampler, m_deviceFrame);
    hipError_t e = hipDeviceSynchronize();
    //CPerformanceMonitoring::endMeasurement("init");
  }

  void Raytracer::freeDeviceMemory() {
    m_scene.freeDeviceMemory();
    hipFree(m_deviceCamera);
    hipFree(m_deviceFrameData);
    hipFree(m_deviceFrame);
    hipFree(m_deviceAverage);
    hipFree(m_deviceTonemappingValue);
  }
  SFrame Raytracer::retrieveFrame() const {
    SFrame frame;
    uint32_t entries = m_frameWidth * m_frameHeight * m_bpp;
    frame.width = m_frameWidth;
    frame.height = m_frameHeight;
    frame.bpp = m_bpp;
    frame.data.resize(entries);
    hipMemcpy(frame.data.data(), m_deviceFrameData, entries * sizeof(float), hipMemcpyDeviceToHost);
    frame.dataBytes.resize(entries);
    hipMemcpy(frame.dataBytes.data(), m_deviceFrameDataBytes, entries * sizeof(uint8_t), hipMemcpyDeviceToHost);
    return frame;
  }
}