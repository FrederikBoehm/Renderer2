#include "backend/build_optix_accel.hpp"
#include <optix/optix_stubs.h>
#include "utility/debugging.hpp"
#include "backend/rt_backend.hpp"

namespace rt {
  void buildOptixAccel(const OptixBuildInput& buildInput, OptixTraversableHandle* traversableHandle, hipDeviceptr_t* deviceGasBuffer) {
    OptixAccelBuildOptions accelOptions = {};
    accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
    accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

    OptixAccelBufferSizes gasBufferSizes;
    const OptixDeviceContext& context = CRTBackend::instance()->context();
    OPTIX_ASSERT(optixAccelComputeMemoryUsage(context, &accelOptions, &buildInput, 1, &gasBufferSizes));

    hipDeviceptr_t d_tempBufferGas;
    CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&d_tempBufferGas), gasBufferSizes.tempSizeInBytes));
    hipDeviceptr_t d_outputBufferGas;
    CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&d_outputBufferGas), gasBufferSizes.outputSizeInBytes));
    hipDeviceptr_t d_compactedSize;
    CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&d_compactedSize), sizeof(size_t)));

    OptixAccelEmitDesc emitProperty = {};
    emitProperty.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
    emitProperty.result = d_compactedSize;


    OPTIX_ASSERT(optixAccelBuild(CRTBackend::instance()->context(),
      0,
      &accelOptions,
      &buildInput,
      1,
      d_tempBufferGas,
      gasBufferSizes.tempSizeInBytes,
      d_outputBufferGas,
      gasBufferSizes.outputSizeInBytes,
      traversableHandle,
      &emitProperty,
      1));

    CUDA_ASSERT(hipStreamSynchronize(0));

    size_t compactedSize;
    CUDA_ASSERT(hipMemcpy(&compactedSize, reinterpret_cast<void*>(emitProperty.result), sizeof(size_t), hipMemcpyDeviceToHost));
    CUDA_ASSERT(hipFree(reinterpret_cast<void*>(d_compactedSize)));
    if (compactedSize < gasBufferSizes.outputSizeInBytes)
    {
      CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(deviceGasBuffer), compactedSize));
      OPTIX_ASSERT(optixAccelCompact(context, 0, *traversableHandle, *deviceGasBuffer, compactedSize, traversableHandle));
      CUDA_ASSERT(hipFree(reinterpret_cast<void*>(d_outputBufferGas)));
    }
    else
    {
      *deviceGasBuffer = d_outputBufferGas;
    }
    CUDA_ASSERT(hipFree(reinterpret_cast<void*>(d_tempBufferGas)));
  }
}