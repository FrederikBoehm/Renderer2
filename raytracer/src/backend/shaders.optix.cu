#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <crt/hip/device_functions.h>
#include "intersect/hit_information.hpp"
#include <optix/optix_device.h>
#include <glm/glm.hpp>
#include "backend/types.hpp"
#include "camera/camera.hpp"
#include "intersect/ray.hpp"
#include "scene/device_scene.hpp"
#include "scene/device_sceneobject.hpp"
#include <stdio.h>
#include "medium/nvdb_medium.hpp"
#include <nanovdb/util/Ray.h>
#include "integrators/path_integrator_impl.hpp"
#include "camera/pixel_sampler.hpp"
#include "mesh/mesh.hpp"


using namespace rt;

__constant__ rt::SLaunchParams params;


extern "C" __global__ void __closesthit__ch() {
  const rt::CDeviceSceneobject** sceneobjectPtr = reinterpret_cast<const rt::CDeviceSceneobject**>(optixGetSbtDataPointer());
  const rt::CDeviceSceneobject* sceneobject = *sceneobjectPtr;

  unsigned int siAdress[2];
  siAdress[0] = optixGetPayload_0();
  siAdress[1] = optixGetPayload_1();

  SInteraction* si;
  memcpy(&si, siAdress, sizeof(SInteraction*));

  //float3 hitPos = { uint_as_float(optixGetAttribute_0()), uint_as_float(optixGetAttribute_1()), uint_as_float(optixGetAttribute_2()) };
  float3 rayOrigin = optixGetWorldRayOrigin();
  float3 rayDirection = optixGetWorldRayDirection();
  float tMax = optixGetRayTmax();
  float3 hitPos = { rayOrigin.x + tMax * rayDirection.x, rayOrigin.y + tMax * rayDirection.y, rayOrigin.z + tMax * rayDirection.z };
  glm::vec3 normal;
  uint3 launchIdx = optixGetLaunchIndex();
  glm::vec2 tc(0.f);
  if (sceneobject->mesh()) {
    const unsigned int primIdx = optixGetPrimitiveIndex();
    const float2       barycentrics = optixGetTriangleBarycentrics();
    const glm::uvec3& triangle = sceneobject->mesh()->ibo()[primIdx];
    const glm::vec3& N0 = sceneobject->mesh()->normals()[triangle.x];
    const glm::vec3& N1 = sceneobject->mesh()->normals()[triangle.y];
    const glm::vec3& N2 = sceneobject->mesh()->normals()[triangle.z];
    //if (barycentrics.x > 1.f || barycentrics.x < 0.f || barycentrics.y > 1.f || barycentrics.y < 0.f) {
    //  printf("Barycentrics: (%f, %f)\n", barycentrics.x, barycentrics.y);
    //}

    normal = (1.f - barycentrics.x - barycentrics.y) * N0 + barycentrics.x * N1 + barycentrics.y * N2;

    const glm::vec2* tcs = sceneobject->mesh()->tcs();
    if (tcs) {
      const glm::vec2& TC0 = sceneobject->mesh()->tcs()[triangle.x];
      const glm::vec2& TC1 = sceneobject->mesh()->tcs()[triangle.y];
      const glm::vec2& TC2 = sceneobject->mesh()->tcs()[triangle.z];

      tc = (1.f - barycentrics.x - barycentrics.y) * TC0 + barycentrics.x * TC1 + barycentrics.y * TC2;
      //if (tc.x < 0.f || tc.x > 1.f || tc.y < 0.f || tc.y > 1.f) {
      //  printf("ch tc: (%f, %f)\n", tc.x, tc.y);
      //}
    }
  }
  else {
    normal = glm::vec3( uint_as_float(optixGetAttribute_3()), uint_as_float(optixGetAttribute_4()), uint_as_float(optixGetAttribute_5()) );
  }

  si->hitInformation.hit = true;
  si->hitInformation.pos = glm::vec3(hitPos.x, hitPos.y, hitPos.z);
  si->hitInformation.normal = glm::normalize(normal);
  si->hitInformation.tc = tc;
  si->hitInformation.t = optixGetRayTmax();
  si->object = sceneobject;
  si->material = sceneobject->material();
  si->medium = sceneobject->medium();

}

extern "C" __global__ void __anyhit__mesh() {
  const rt::CDeviceSceneobject** sceneobjectPtr = reinterpret_cast<const rt::CDeviceSceneobject**>(optixGetSbtDataPointer());
  const rt::CDeviceSceneobject* sceneobject = *sceneobjectPtr;

  const unsigned int primIdx = optixGetPrimitiveIndex();
  const float2       barycentrics = optixGetTriangleBarycentrics();
  const glm::uvec3& triangle = sceneobject->mesh()->ibo()[primIdx];

  const glm::vec2* tcs = sceneobject->mesh()->tcs();
  if (tcs) {
    const glm::vec2& TC0 = sceneobject->mesh()->tcs()[triangle.x];
    const glm::vec2& TC1 = sceneobject->mesh()->tcs()[triangle.y];
    const glm::vec2& TC2 = sceneobject->mesh()->tcs()[triangle.z];

    glm::vec2 tc = (1.f - barycentrics.x - barycentrics.y) * TC0 + barycentrics.x * TC1 + barycentrics.y * TC2;

    if (!sceneobject->material()->opaque(tc)) {
      optixIgnoreIntersection();
    }
  }
}



extern "C" __global__ void __miss__ms() {
  unsigned int siAdress[2];
  siAdress[0] = optixGetPayload_0();
  siAdress[1] = optixGetPayload_1();

  SInteraction* si;
  memcpy(&si, siAdress, sizeof(SInteraction*));
  si->hitInformation.hit = false;
  si->object = nullptr;
  si->material = nullptr;
  si->medium = nullptr;
}


extern "C" __global__ void __intersection__surface() {
  const rt::CDeviceSceneobject** sceneobjectPtr = reinterpret_cast<const rt::CDeviceSceneobject**>(optixGetSbtDataPointer());
  const rt::CDeviceSceneobject* sceneobject = *sceneobjectPtr;

  float3 tempOrigin = optixGetWorldRayOrigin();
  float3 tempDirection = optixGetWorldRayDirection();
  rt::CRay ray(glm::vec3(tempOrigin.x, tempOrigin.y, tempOrigin.z), glm::vec3(tempDirection.x, tempDirection.y, tempDirection.z), optixGetRayTmax());
  rt::SInteraction si = sceneobject->intersect(ray);

  if (si.hitInformation.hit) {
    optixReportIntersection(si.hitInformation.t,
      0,
      float_as_uint(si.hitInformation.pos.x),
      float_as_uint(si.hitInformation.pos.y),
      float_as_uint(si.hitInformation.pos.z),
      float_as_uint(si.hitInformation.normal.x),
      float_as_uint(si.hitInformation.normal.y),
      float_as_uint(si.hitInformation.normal.z));
  }
}


extern "C" __global__ void __intersection__volume() {
  uint3 launchIdx = optixGetLaunchIndex();
  uint3 launchDim = optixGetLaunchDimensions();
  uint32_t samplerId = launchIdx.y * launchDim.x + launchIdx.x;
  CSampler& sampler = params.sampler[samplerId];
  const rt::CDeviceSceneobject** sceneobjectPtr = reinterpret_cast<const rt::CDeviceSceneobject**>(optixGetSbtDataPointer());
  const rt::CDeviceSceneobject* sceneobject = *sceneobjectPtr;

  float3 tempOrigin = optixGetWorldRayOrigin();
  float3 tempDirection = optixGetWorldRayDirection();
  glm::vec3 rayOrigin(tempOrigin.x, tempOrigin.y, tempOrigin.z);
  glm::vec3 rayDirection(tempDirection.x, tempDirection.y, tempDirection.z);

  const CNVDBMedium* medium = static_cast<const CNVDBMedium*>(sceneobject->medium());
  const nanovdb::BBox<nanovdb::Vec3R>& aabb = medium->grid()->worldBBox();
  float t0 = 0.f;
  float t1 = CRay::DEFAULT_TMAX;
  float initialT1 = t1;

  nanovdb::Ray<float> ray = nanovdb::Ray<float>(reinterpret_cast<const nanovdb::Vec3f&>(tempOrigin), reinterpret_cast<const nanovdb::Vec3f&>(tempDirection), t0, t1);
  bool intersects = ray.intersects(aabb, t0, t1);
  if (intersects && t1 < initialT1) {
    float t;
    nanovdb::Vec3R nanovdbOrigin(tempOrigin.x, tempOrigin.y, tempOrigin.z);
    if (aabb.isInside(nanovdbOrigin)) {// When ray starts in medium it seems that t1 gives first intersection while t0 remaining unchanged
      t = t1;
    }
    else {
      t = t0;
    }

    glm::vec3 intersectionPos = rayOrigin + t * rayDirection;
    optixReportIntersection(
      t,
      0,
      float_as_uint(intersectionPos.x),
      float_as_uint(intersectionPos.y),
      float_as_uint(intersectionPos.z),
      float_as_uint(0.f),
      float_as_uint(0.f),
      float_as_uint(0.f)
    );

  }
}



extern "C" __global__ void __raygen__rg() {
  uint3 launchIdx = optixGetLaunchIndex();
  uint3 launchDim = optixGetLaunchDimensions();

  uint32_t samplerId = launchIdx.y * launchDim.x + launchIdx.x;

  CPixelSampler pixelSampler(params.camera, launchIdx.x, launchIdx.y, &(params.sampler[samplerId]));
  //CPathIntegrator integrator((CDeviceScene*)sharedScene, &pixelSampler, &(sampler[samplerId]), numSamples);
  CPathIntegrator integrator(params.scene, &pixelSampler, &(params.sampler[samplerId]), params.numSamples);
  glm::vec3 L = integrator.Li();

  uint32_t currentPixel = params.bpp * (launchIdx.y * launchDim.x + launchIdx.x);

  params.data[currentPixel + 0] += L.x;
  params.data[currentPixel + 1] += L.y;
  params.data[currentPixel + 2] += L.z;

}
