#include "hip/hip_runtime.h"
#include "integrators/direct_lighting_integrator.hpp"
#include "sampling/sampler.hpp"
#include "scene/scene.hpp"
#include "camera/pixel_sampler.hpp"
#include <cmath>
#include "sampling/mis.hpp"

namespace rt {
  CDirectLightingIntegrator::CDirectLightingIntegrator(CDeviceScene* scene, CPixelSampler* pixelSampler, CSampler* sampler, uint16_t numSamples):
    m_scene(scene),
    m_pixelSampler(pixelSampler),
    m_sampler(sampler),
    m_numSamples(numSamples) {

  }

  glm::vec3 CDirectLightingIntegrator::Li(EIntegrationStrategy strategy) {
    glm::vec3 L(0.0f);
    if (strategy == UNIFORM_SAMPLE_HEMISPHERE) {
      Ray eyeRay = m_pixelSampler->samplePixel();

      SSurfaceInteraction si = m_scene->intersect(eyeRay);
      if (si.hitInformation.hit) {
        if (si.material.Le() != glm::vec3(0.0f)) { // Hit on light source
          L = si.material.Le() / (float)m_numSamples;
        }
        else {
          glm::vec3 tangentSpaceDirection = m_sampler->uniformSampleHemisphere();
          // Construct tangent space
          glm::vec3 notN = glm::normalize(glm::vec3(si.hitInformation.normal.x + 1.0f, si.hitInformation.normal.x + 2.0f, si.hitInformation.normal.x + 3.0f));
          glm::vec3 tangent = glm::normalize(glm::cross(notN, si.hitInformation.normal));
          glm::vec3 bitangent = glm::normalize(glm::cross(si.hitInformation.normal, tangent));

          glm::mat4 tangentToWorld(glm::vec4(tangent, 0.0f), glm::vec4(bitangent, 0.0f), glm::vec4(si.hitInformation.normal, 0.0f), glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
          glm::mat4 worldToTangent = glm::inverse(tangentToWorld);
          glm::vec3 worldSpaceDirection = glm::normalize(glm::vec3(tangentToWorld * glm::vec4(tangentSpaceDirection, 0.0f)));

          Ray shadowRay = Ray(si.hitInformation.pos + 1.0e-6f * si.hitInformation.normal, worldSpaceDirection);
          SSurfaceInteraction si2 = m_scene->intersect(shadowRay);

          Ray eyeRayTangent = eyeRay.transform(worldToTangent);

          glm::vec3 f = si.material.f(-eyeRayTangent.m_direction, tangentSpaceDirection);
          glm::vec3 Le = si2.material.Le();
          float cosine = glm::max(glm::dot(si.hitInformation.normal, shadowRay.m_direction), 0.0f);
          float pdf = m_sampler->uniformHemispherePdf();

          L = f * Le * cosine / ((float)m_numSamples * pdf);
        }
      }
    }
    else if (strategy == IMPORTANCE_SAMPLE_LIGHTSOURCES) {
      Ray eyeRay = m_pixelSampler->samplePixel();

      SSurfaceInteraction si = m_scene->intersect(eyeRay);
      if (si.hitInformation.hit) {
        if (si.material.Le() != glm::vec3(0.0f)) { // Hit on light source
          L = si.material.Le() / (float)m_numSamples;
        }
        else {
          // Construct tangent space
          glm::vec3 notN = glm::normalize(glm::vec3(si.hitInformation.normal.x + 1.0f, si.hitInformation.normal.x + 2.0f, si.hitInformation.normal.x + 3.0f));
          glm::vec3 tangent = glm::normalize(glm::cross(notN, si.hitInformation.normal));
          glm::vec3 bitangent = glm::normalize(glm::cross(si.hitInformation.normal, tangent));

          glm::mat4 tangentToWorld(glm::vec4(tangent, 0.0f), glm::vec4(bitangent, 0.0f), glm::vec4(si.hitInformation.normal, 0.0f), glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
          glm::mat4 worldToTangent = glm::inverse(tangentToWorld);

          float lightsPdf;
          glm::vec3 pos = m_scene->sampleLightSources(*m_sampler, &lightsPdf);
          glm::vec3 worldSpaceDirection = glm::normalize(m_scene->sampleLightSources(*m_sampler, &lightsPdf) - si.hitInformation.pos);
          glm::vec3 tangentSpaceDirection = glm::normalize(glm::vec3(worldToTangent * glm::vec4(worldSpaceDirection, 0.0f)));

          Ray shadowRay = Ray(si.hitInformation.pos + 1.0e-6f * si.hitInformation.normal, worldSpaceDirection);
          SSurfaceInteraction si2 = m_scene->intersect(shadowRay);

          Ray eyeRayTangent = eyeRay.transform(worldToTangent);

          float lightSamplingPdf = m_scene->lightSourcePdf(si2, shadowRay);
          glm::vec3 f = si.material.f(-eyeRayTangent.m_direction, tangentSpaceDirection);
          glm::vec3 Le = si2.material.Le();
          float distance = glm::length(si.hitInformation.pos - si2.hitInformation.pos);
          float G = glm::max(glm::dot(si.hitInformation.normal, shadowRay.m_direction), 0.0f) * glm::max(glm::dot(si2.hitInformation.normal, -shadowRay.m_direction), 0.0f) / (distance * distance);

          L = f * Le * G / ((float)m_numSamples * lightsPdf * lightSamplingPdf);
        }
      }
    }
    else if (strategy == IMPORTANCE_SAMPLE_COSINE) {
      Ray eyeRay = m_pixelSampler->samplePixel();

      SSurfaceInteraction si = m_scene->intersect(eyeRay);
      if (si.hitInformation.hit) {
        if (si.material.Le() != glm::vec3(0.0f)) { // Hit on light source
          L = si.material.Le() / (float)m_numSamples;
        }
        else {
          glm::vec3 tangentSpaceDirection = m_sampler->cosineSampleHemisphere();
          // Construct tangent space
          glm::vec3 notN = glm::normalize(glm::vec3(si.hitInformation.normal.x + 1.0f, si.hitInformation.normal.x + 2.0f, si.hitInformation.normal.x + 3.0f));
          glm::vec3 tangent = glm::normalize(glm::cross(notN, si.hitInformation.normal));
          glm::vec3 bitangent = glm::normalize(glm::cross(si.hitInformation.normal, tangent));

          glm::mat4 tangentToWorld(glm::vec4(tangent, 0.0f), glm::vec4(bitangent, 0.0f), glm::vec4(si.hitInformation.normal, 0.0f), glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
          glm::mat4 worldToTangent = glm::inverse(tangentToWorld);
          glm::vec3 worldSpaceDirection = glm::normalize(glm::vec3(tangentToWorld * glm::vec4(tangentSpaceDirection, 0.0f)));

          Ray shadowRay = Ray(si.hitInformation.pos + 1.0e-6f * si.hitInformation.normal, worldSpaceDirection);
          SSurfaceInteraction si2 = m_scene->intersect(shadowRay);

          Ray eyeRayTangent = eyeRay.transform(worldToTangent);

          glm::vec3 f = si.material.f(-eyeRayTangent.m_direction, tangentSpaceDirection);
          glm::vec3 Le = si2.material.Le();
          float cosine = glm::max(glm::dot(si.hitInformation.normal, shadowRay.m_direction), 0.0f);
          float pdf = m_sampler->cosineHemispherePdf(cosine);

          L = f * Le * cosine / ((float)m_numSamples * pdf);
        }
      }
    }
    else if (strategy == IMPORTANCE_SAMPLE_BRDF) {
      Ray eyeRay = m_pixelSampler->samplePixel();

      SSurfaceInteraction si = m_scene->intersect(eyeRay);
      if (si.hitInformation.hit) {
        if (si.material.Le() != glm::vec3(0.0f)) { // Hit on light source
          L = si.material.Le() / (float)m_numSamples;
        }
        else {
          // Construct tangent space
          glm::vec3 notN = glm::normalize(glm::vec3(si.hitInformation.normal.x + 1.0f, si.hitInformation.normal.x + 2.0f, si.hitInformation.normal.x + 3.0f));
          glm::vec3 tangent = glm::normalize(glm::cross(notN, si.hitInformation.normal));
          glm::vec3 bitangent = glm::normalize(glm::cross(si.hitInformation.normal, tangent));

          glm::mat4 tangentToWorld(glm::vec4(tangent, 0.0f), glm::vec4(bitangent, 0.0f), glm::vec4(si.hitInformation.normal, 0.0f), glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
          glm::mat4 worldToTangent = glm::inverse(tangentToWorld);


          Ray eyeRayTangent = eyeRay.transform(worldToTangent);
          glm::vec3 wi;
          float pdf;
          //glm::vec3 f = si.material.sampleF(-eyeRayTangent.m_direction, &wi, *m_sampler, &pdf);
          si.material.sampleF(-eyeRayTangent.m_direction, &wi, *m_sampler, &pdf);
          glm::vec3 f = si.material.f(-eyeRayTangent.m_direction, wi);
          
          glm::vec3 worldSpaceDirection = glm::normalize(glm::vec3(tangentToWorld * glm::vec4(wi, 0.0f)));
          Ray shadowRay = Ray(si.hitInformation.pos + 1.0e-6f * si.hitInformation.normal, worldSpaceDirection);
          SSurfaceInteraction si2 = m_scene->intersect(shadowRay);

          glm::vec3 Le = si2.material.Le();
          float cosine = glm::max(glm::dot(si.hitInformation.normal, shadowRay.m_direction), 0.0f);

          L = f * Le * cosine / ((float)m_numSamples * pdf);
        }
      }
    }
    else if (strategy == MULTIPLE_IMPORTANCE_SAMPLE) {
      Ray eyeRay = m_pixelSampler->samplePixel();

      SSurfaceInteraction si = m_scene->intersect(eyeRay);
      if (si.hitInformation.hit) {
        if (si.material.Le() != glm::vec3(0.0f)) { // Hit on light source
          L = si.material.Le() / (float)m_numSamples;
        }
        else {
          // Construct tangent space
          glm::vec3 notN = glm::normalize(glm::vec3(si.hitInformation.normal.x + 1.0f, si.hitInformation.normal.x + 2.0f, si.hitInformation.normal.x + 3.0f));
          glm::vec3 tangent = glm::normalize(glm::cross(notN, si.hitInformation.normal));
          glm::vec3 bitangent = glm::normalize(glm::cross(si.hitInformation.normal, tangent));

          glm::mat4 tangentToWorld(glm::vec4(tangent, 0.0f), glm::vec4(bitangent, 0.0f), glm::vec4(si.hitInformation.normal, 0.0f), glm::vec4(0.0f, 0.0f, 0.0f, 1.0f));
          glm::mat4 worldToTangent = glm::inverse(tangentToWorld);

          // Sample light source
          float lightsPdf = 0.0f;
          //glm::vec3 lightPos = m_scene->sampleLightSources(*m_sampler, &lightsPdf);
          glm::vec3 lightWorldSpaceDirection = glm::normalize(m_scene->sampleLightSources(*m_sampler, &lightsPdf) - si.hitInformation.pos);
          glm::vec3 lightTangentSpaceDirection = glm::normalize(glm::vec3(worldToTangent * glm::vec4(lightWorldSpaceDirection, 0.0f)));

          Ray shadowRayLight = Ray(si.hitInformation.pos + 1.0e-6f * si.hitInformation.normal, lightWorldSpaceDirection);
          SSurfaceInteraction si2 = m_scene->intersect(shadowRayLight);

          Ray eyeRayTangent = eyeRay.transform(worldToTangent);

          float lightSamplingPdf = m_scene->lightSourcePdf(si2, shadowRayLight);
          float lightPdf = lightsPdf * lightSamplingPdf;
          float brdfPdf = si.material.pdf(-eyeRayTangent.m_direction, lightTangentSpaceDirection);
          float mis_weight = balanceHeuristic(1, lightsPdf, 1, brdfPdf);

          glm::vec3 f = si.material.f(-eyeRayTangent.m_direction, lightTangentSpaceDirection);
          glm::vec3 Le = si2.material.Le();
          float distance = glm::length(si.hitInformation.pos - si2.hitInformation.pos);
          float G = glm::max(glm::dot(si.hitInformation.normal, shadowRayLight.m_direction), 0.0f) * glm::max(glm::dot(si2.hitInformation.normal, -shadowRayLight.m_direction), 0.0f) / (distance * distance);

          L += mis_weight * f * Le * G / ((float)m_numSamples * lightPdf);

          // Sample BRDF
          glm::vec3 wi(0.0f);
          f = si.material.sampleF(-eyeRayTangent.m_direction, &wi, *m_sampler, &brdfPdf);
          glm::vec3 brdfWorldSpaceDirection = glm::normalize(glm::vec3(tangentToWorld * glm::vec4(wi, 0.0f)));
          Ray shadowRayBrdf = Ray(si.hitInformation.pos + 1.0e-6f * si.hitInformation.normal, brdfWorldSpaceDirection);
          si2 = m_scene->intersect(shadowRayBrdf);

          Le = si2.material.Le();
          if (Le != glm::vec3(0.0f)) {
            float cosine = glm::max(glm::dot(si.hitInformation.normal, shadowRayBrdf.m_direction), 0.0f);
            lightSamplingPdf = m_scene->lightSourcePdf(si2, shadowRayBrdf);
            lightsPdf = m_scene->lightSourcesPdf(si2);
            lightPdf = lightSamplingPdf * lightsPdf;
            mis_weight = balanceHeuristic(1, brdfPdf, 1, lightPdf);

            L += mis_weight * f * Le * cosine / ((float)m_numSamples * brdfPdf);
          }
          
        }
      }
    }
    return L;
  }
}