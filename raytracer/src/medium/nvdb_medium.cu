#include "hip/hip_runtime.h"
#include "medium/nvdb_medium.hpp"
#include <nanovdb/NanoVDB.h>
#include "utility/functions.hpp"
#include "intersect/ray.hpp"
#include "sampling/sampler.hpp"
#include "scene/interaction.hpp"
#include <glm/gtx/transform.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "medium/sggx_phase_function.hpp"
#include "utility/debugging.hpp"
#include "backend/rt_backend.hpp"
#include <optix/optix_stubs.h>
#include "medium/phase_function_impl.hpp"
#include "medium/medium_impl.hpp"
#include "backend/build_optix_accel.hpp"

namespace rt {
  CNVDBMedium::CNVDBMedium(const std::string& path, const glm::vec3& sigma_a, const glm::vec3& sigma_s, float g, const glm::vec3& worldPos, const glm::vec3& n, const glm::vec3& scaling):
    CMedium(EMediumType::NVDB_MEDIUM),
    m_isHostObject(true),
    m_handle(getHandle(path)),
    m_grid(m_handle->grid<float>()),
    m_readAccessor(new nanovdb::DefaultReadAccessor<float>(m_grid->getAccessor())),
    m_deviceAabb(NULL),
    m_size(getMediumSize(m_grid->worldBBox(), m_grid->voxelSize())),
    m_sigma_a(sigma_a),
    m_sigma_s(sigma_s),
    m_phase(new CHenyeyGreensteinPhaseFunction(g)),
    m_sigma_t(sigma_a.z + sigma_s.z),
    m_invMaxDensity(1.f / getMaxValue(m_grid)),
    m_deviceResource(nullptr) {
    const nanovdb::CoordBBox box = m_grid->indexBBox();
    if (m_grid->activeVoxelCount() == 0) {
      m_ibbMin = glm::ivec3(0);
      m_ibbMax = glm::ivec3(0);
    }
    else {
      m_ibbMin = glm::ivec3(box.min().x(), box.min().y(), box.min().z());
      m_ibbMax = glm::ivec3(box.max().x(), box.max().y(), box.max().z());
    }
    nanovdb::BBoxR worldBB = m_grid->worldBBox();
    m_indexToModel = getIndexToModelTransformation(m_grid->map(), m_ibbMin, m_size);
    m_worldBB = worldBB;
    m_modelToIndex = glm::inverse(m_indexToModel);
  }

  CNVDBMedium::CNVDBMedium(const std::string& path, const glm::vec3& sigma_a, const glm::vec3& sigma_s, const SSGGXDistributionParameters& sggxDiffuse, const SSGGXDistributionParameters& sggxSpecular, const glm::vec3& worldPos, const glm::vec3& n, const glm::vec3& scaling) :
    CMedium(EMediumType::NVDB_MEDIUM),
    m_isHostObject(true),
    m_handle(getHandle(path)),
    m_grid(m_handle->grid<float>()),
    m_readAccessor(new nanovdb::DefaultReadAccessor<float>(m_grid->getAccessor())),
    m_deviceAabb(NULL),
    m_size(getMediumSize(m_grid->worldBBox(), m_grid->voxelSize())),
    m_sigma_a(sigma_a),
    m_sigma_s(sigma_s),
    m_phase(new CSGGXPhaseFunction(sggxDiffuse, sggxSpecular)),
    m_sigma_t(sigma_a.z + sigma_s.z),
    m_invMaxDensity(1.f / getMaxValue(m_grid)),
    m_deviceResource(nullptr) {
    const nanovdb::CoordBBox box = m_grid->indexBBox();
    if (m_grid->activeVoxelCount() == 0) {
      m_ibbMin = glm::ivec3(0);
      m_ibbMax = glm::ivec3(0);
    }
    else {
      m_ibbMin = glm::ivec3(box.min().x(), box.min().y(), box.min().z());
      m_ibbMax = glm::ivec3(box.max().x(), box.max().y(), box.max().z());
    }
    nanovdb::BBoxR worldBB = m_grid->worldBBox();
    m_indexToModel = getIndexToModelTransformation(m_grid->map(), m_ibbMin, m_size);
    m_worldBB = worldBB;
    m_modelToIndex = glm::inverse(m_indexToModel);
  }

  CNVDBMedium::CNVDBMedium(const std::string& path, const glm::vec3& sigma_a, const glm::vec3& sigma_s, float diffuseRoughness, float specularRoughness) :
    CMedium(EMediumType::NVDB_MEDIUM),
    m_pathLength(path.size()),
    m_path((char*)malloc(path.size())),
    m_isHostObject(true),
    m_handle(getHandle(path)),
    m_grid(m_handle->grid<float>()),
    m_readAccessor(new nanovdb::DefaultReadAccessor<float>(m_grid->getAccessor())),
    m_deviceAabb(NULL),
    m_size(getMediumSize(m_grid->worldBBox(), m_grid->voxelSize())),
    m_sigma_a(sigma_a),
    m_sigma_s(sigma_s),
    m_phase(new CSGGXPhaseFunction(diffuseRoughness, specularRoughness)),
    m_sigma_t(sigma_a.z + sigma_s.z),
    m_invMaxDensity(1.f / getMaxValue(m_grid)),
    m_deviceGasBuffer(NULL),
    m_deviceResource(nullptr) {

    memcpy(m_path, path.data(), path.size());

    auto worldBBDim = m_grid->worldBBox().dim();
    auto voxelSize = m_grid->voxelSize();
    auto voxelCount = m_grid->activeVoxelCount();
    const nanovdb::CoordBBox box = m_grid->indexBBox();
    if (m_grid->activeVoxelCount() == 0) {
      m_ibbMin = glm::ivec3(0);
      m_ibbMax = glm::ivec3(0);
    }
    else {
      m_ibbMin = glm::ivec3(box.min().x(), box.min().y(), box.min().z());
      m_ibbMax = glm::ivec3(box.max().x(), box.max().y(), box.max().z());
    }
    m_indexToModel = getIndexToModelTransformation(m_grid->map(), m_ibbMin, m_size);
    m_worldBB = m_grid->worldBBox();
    m_modelToIndex = glm::inverse(m_indexToModel);
  }

  CNVDBMedium::CNVDBMedium() :
    CMedium(EMediumType::NVDB_MEDIUM),
    m_pathLength(0),
    m_path(nullptr),
    m_isHostObject(true),
    m_handle(nullptr),
    m_grid(nullptr),
    m_readAccessor(nullptr),
    m_worldBB(),
    m_deviceAabb(NULL),
    m_size(0),
    m_indexToModel(1.f),
    m_modelToIndex(1.f),
    m_sigma_a(0.f),
    m_sigma_s(0.f),
    m_phase(nullptr),
    m_ibbMin(0),
    m_ibbMax(0),
    m_sigma_t(0.f),
    m_invMaxDensity(0.f),
    m_deviceResource(nullptr) {

  }

  CNVDBMedium::CNVDBMedium(CNVDBMedium&& medium) :
    CMedium(std::move(medium.type())),
    m_pathLength(std::move(medium.m_pathLength)),
    m_path(std::exchange(medium.m_path, nullptr)),
    m_isHostObject(std::move(medium.m_isHostObject)),
    m_handle(std::exchange(medium.m_handle, nullptr)),
    m_grid(std::exchange(medium.m_grid, nullptr)),
    m_readAccessor(std::exchange(medium.m_readAccessor, nullptr)),
    m_worldBB(std::move(medium.m_worldBB)),
    m_deviceAabb(std::exchange(medium.m_deviceAabb, NULL)),
    m_size(std::move(medium.m_size)),
    m_indexToModel(std::move(medium.m_indexToModel)),
    m_modelToIndex(std::move(medium.m_modelToIndex)),
    m_sigma_a(std::move(medium.m_sigma_a)),
    m_sigma_s(std::move(medium.m_sigma_s)),
    m_phase(std::exchange(medium.m_phase, nullptr)),
    m_ibbMin(std::move(medium.m_ibbMin)),
    m_ibbMax(std::move(medium.m_ibbMax)),
    m_sigma_t(std::move(medium.m_sigma_t)),
    m_invMaxDensity(std::move(medium.m_invMaxDensity)),
    m_deviceResource(std::exchange(medium.m_deviceResource, nullptr)) {
  }

  CNVDBMedium::~CNVDBMedium() {
    if (m_isHostObject) {
      delete m_path;
      delete m_readAccessor;
      delete m_handle;
      delete m_phase;
    }
  }


  CNVDBMedium& CNVDBMedium::operator=(const CNVDBMedium&& medium) {
    return *this;
  }

  

  void CNVDBMedium::allocateDeviceMemory() {
    if (m_deviceResource) {
      freeDeviceMemory();
      delete m_deviceResource;
    }

    m_deviceResource = new DeviceResource();
    CUDA_ASSERT(hipMalloc(&m_deviceResource->d_readAccessor, sizeof(nanovdb::DefaultReadAccessor<float>)));
    switch (m_phase->type()) {
    case EPhaseFunction::HENYEY_GREENSTEIN:
      CUDA_ASSERT(hipMalloc(&m_deviceResource->d_phase, sizeof(CHenyeyGreensteinPhaseFunction)));
      break;
    case EPhaseFunction::SGGX:
      CUDA_ASSERT(hipMalloc(&m_deviceResource->d_phase, sizeof(CSGGXPhaseFunction)));
      break;
    }
  }

  CNVDBMedium CNVDBMedium::copyToDevice() const {
    m_handle->deviceUpload();

    
    CNVDBMedium medium;
    medium.m_isHostObject = false;
    medium.m_handle = this->m_handle;
    medium.m_grid = m_handle->deviceGrid<float>();
    if (!medium.m_grid) {
      fprintf(stderr, "GridHandle does not contain a valid device grid");
    }
    if (m_deviceResource) {
      medium.m_readAccessor = m_deviceResource->d_readAccessor;
      CUDA_ASSERT(hipMemcpy(m_deviceResource->d_readAccessor, this->m_readAccessor, sizeof(nanovdb::DefaultReadAccessor<float>), hipMemcpyHostToDevice));

      medium.m_phase = m_deviceResource->d_phase;
      switch (m_phase->type()) {
      case EPhaseFunction::HENYEY_GREENSTEIN:
        CUDA_ASSERT(hipMemcpy(m_deviceResource->d_phase, this->m_phase, sizeof(CHenyeyGreensteinPhaseFunction), hipMemcpyHostToDevice));
        break;
      case EPhaseFunction::SGGX:
        CUDA_ASSERT(hipMemcpy(m_deviceResource->d_phase, this->m_phase, sizeof(CSGGXPhaseFunction), hipMemcpyHostToDevice));
        break;
      }
    }
    else {
      medium.m_readAccessor = nullptr;
      fprintf(stderr, "No device resource for CNVDBMedium");
    }
    medium.m_size = this->m_size;
    medium.m_indexToModel = this->m_indexToModel;
    medium.m_modelToIndex = this->m_modelToIndex;
    medium.m_sigma_a = this->m_sigma_a;
    medium.m_sigma_s = this->m_sigma_s;
    medium.m_sigma_t = this->m_sigma_t;
    medium.m_invMaxDensity = this->m_invMaxDensity;
    medium.m_deviceResource = nullptr;
    medium.m_ibbMin = m_ibbMin;
    medium.m_ibbMax = m_ibbMax;
    medium.m_worldBB = m_worldBB;
    
    return medium;
  }

  void CNVDBMedium::freeDeviceMemory() const {
    if (m_deviceResource) {
      CUDA_ASSERT(hipFree(m_deviceResource->d_readAccessor));
      CUDA_ASSERT(hipFree(m_deviceResource->d_phase));
    }
    CUDA_ASSERT(hipFree(reinterpret_cast<void*>(m_deviceAabb)));
    CUDA_ASSERT(hipFree(reinterpret_cast<void*>(m_deviceGasBuffer)));
  }

  void CNVDBMedium::buildOptixAccel() {
    if (!m_deviceAabb) {
      CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&m_deviceAabb), sizeof(OptixAabb)));
      CUDA_ASSERT(hipMemcpy(reinterpret_cast<void*>(m_deviceAabb), &m_worldBB, sizeof(OptixAabb), hipMemcpyHostToDevice));
    }

    OptixBuildInput buildInput;
    buildInput = {};
    buildInput.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
    buildInput.customPrimitiveArray.aabbBuffers = &m_deviceAabb;
    OptixGeometryFlags flags[] = { OPTIX_GEOMETRY_FLAG_NONE };
    buildInput.customPrimitiveArray.flags = reinterpret_cast<unsigned int*>(flags);
    buildInput.customPrimitiveArray.numSbtRecords = 1;
    buildInput.customPrimitiveArray.numPrimitives = 1;
    buildInput.customPrimitiveArray.sbtIndexOffsetBuffer = 0;
    buildInput.customPrimitiveArray.sbtIndexOffsetSizeInBytes = 0;
    buildInput.customPrimitiveArray.primitiveIndexOffset = 0;
    rt::buildOptixAccel(buildInput, &m_traversableHandle, &m_deviceGasBuffer);
  }

  glm::ivec3 CNVDBMedium::getMediumSize(const nanovdb::BBox<nanovdb::Vec3R>& boundingBox, const nanovdb::Vec3R& voxelSize) {
    nanovdb::Vec3R size = boundingBox.dim() / voxelSize;
    return glm::ivec3(size[0], size[1], size[2]);
  }

  float CNVDBMedium::getMaxValue(const nanovdb::NanoGrid<float>* grid) {
    float min = 0.f;
    float max = 0.f;
    grid->tree().extrema(min, max);
    return max;
  }

  glm::mat4 CNVDBMedium::getIndexToModelTransformation(const nanovdb::Map& map, const glm::ivec3& ibbMin, const glm::ivec3& size) {
    glm::mat4 nanoIndexToWorld(map.mMatF[0], map.mMatF[3], map.mMatF[6], 0.f,
                               map.mMatF[1], map.mMatF[4], map.mMatF[7], 0.f,
                               map.mMatF[2], map.mMatF[5], map.mMatF[8], 0.f,
                               map.mVecF[0], map.mVecF[1], map.mVecF[2], 1.f); // [IdxMin, IdxMax] to world space
    glm::mat4 indexToNano((float)size[0], 0.f, 0.f, 0.f,
                          0.f, (float)size[1], 0.f, 0.f,
                          0.f, 0.f, (float)size[2], 0.f,
                          ibbMin.x, ibbMin.y, ibbMin.z, 1.f); // [0, 1] to [IdxMin, IdxMax] (Nanovdb index space
    return nanoIndexToWorld * indexToNano;
  }

  nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>* CNVDBMedium::getHandle(const std::string& path) {
    nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>* handle = nullptr;
    try {
      handle = new nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>(nanovdb::io::readGrid<nanovdb::CudaDeviceBuffer>(path));
    }
    catch (const std::exception& e) {
      fprintf(stderr, "Couldn't load nvdb file: %s", e.what());
      handle = nullptr;
    }
    return handle;
  }

  OptixProgramGroup CNVDBMedium::getOptixProgramGroup() const {
    return CRTBackend::instance()->programGroups().m_hitVolume;
  }

  std::string CNVDBMedium::path() const {
    return std::string(m_path, m_pathLength);
  }

  OptixTraversableHandle CNVDBMedium::getOptixHandle() const {
    return m_traversableHandle;
  }
}