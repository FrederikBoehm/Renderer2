#include "hip/hip_runtime.h"
#include "medium/nvdb_medium.hpp"
#include <nanovdb/NanoVDB.h>
#include "utility/functions.hpp"
#include "intersect/ray.hpp"
#include "sampling/sampler.hpp"
#include "scene/interaction.hpp"
#include <glm/gtx/transform.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "medium/sggx_phase_function.hpp"
#include "utility/debugging.hpp"
#include "backend/rt_backend.hpp"
#include <optix/optix_stubs.h>
#include "medium/phase_function_impl.hpp"
#include "medium/medium_impl.hpp"

namespace rt {
  CNVDBMedium::CNVDBMedium(const std::string& path, const glm::vec3& sigma_a, const glm::vec3& sigma_s, float g, const glm::vec3& worldPos, const glm::vec3& n, const glm::vec3& scaling):
    CMedium(EMediumType::NVDB_MEDIUM),
    m_isHostObject(true),
    m_handle(getHandle(path)),
    m_grid(m_handle->grid<float>()),
    m_readAccessor(new nanovdb::DefaultReadAccessor<float>(m_grid->getAccessor())),
    m_deviceAabb(NULL),
    m_size(getMediumSize(m_grid->worldBBox(), m_grid->voxelSize())),
    m_sigma_a(sigma_a),
    m_sigma_s(sigma_s),
    m_phase(new CHenyeyGreensteinPhaseFunction(g)),
    m_sigma_t(sigma_a.z + sigma_s.z),
    m_invMaxDensity(1.f / getMaxValue(m_grid)),
    m_deviceResource(nullptr) {
    const nanovdb::CoordBBox box = m_grid->indexBBox();
    if (m_grid->activeVoxelCount() == 0) {
      m_ibbMin = glm::ivec3(0);
      m_ibbMax = glm::ivec3(0);
    }
    else {
      m_ibbMin = glm::ivec3(box.min().x(), box.min().y(), box.min().z());
      m_ibbMax = glm::ivec3(box.max().x(), box.max().y(), box.max().z());
    }
    nanovdb::BBoxR worldBB = m_grid->worldBBox();
    m_mediumToWorld = getMediumToWorldTransformation(m_grid->map(), m_ibbMin, m_size, worldPos, n, scaling, &worldBB);
    m_worldBB = worldBB;
    m_worldToMedium = glm::inverse(m_mediumToWorld);
  }

  CNVDBMedium::CNVDBMedium(const std::string& path, const glm::vec3& sigma_a, const glm::vec3& sigma_s, const SSGGXDistributionParameters& sggxDiffuse, const SSGGXDistributionParameters& sggxSpecular, const glm::vec3& worldPos, const glm::vec3& n, const glm::vec3& scaling) :
    CMedium(EMediumType::NVDB_MEDIUM),
    m_isHostObject(true),
    m_handle(getHandle(path)),
    m_grid(m_handle->grid<float>()),
    m_readAccessor(new nanovdb::DefaultReadAccessor<float>(m_grid->getAccessor())),
    m_deviceAabb(NULL),
    m_size(getMediumSize(m_grid->worldBBox(), m_grid->voxelSize())),
    m_sigma_a(sigma_a),
    m_sigma_s(sigma_s),
    m_phase(new CSGGXPhaseFunction(sggxDiffuse, sggxSpecular)),
    m_sigma_t(sigma_a.z + sigma_s.z),
    m_invMaxDensity(1.f / getMaxValue(m_grid)),
    m_deviceResource(nullptr) {
    const nanovdb::CoordBBox box = m_grid->indexBBox();
    if (m_grid->activeVoxelCount() == 0) {
      m_ibbMin = glm::ivec3(0);
      m_ibbMax = glm::ivec3(0);
    }
    else {
      m_ibbMin = glm::ivec3(box.min().x(), box.min().y(), box.min().z());
      m_ibbMax = glm::ivec3(box.max().x(), box.max().y(), box.max().z());
    }
    nanovdb::BBoxR worldBB = m_grid->worldBBox();
    m_mediumToWorld = getMediumToWorldTransformation(m_grid->map(), m_ibbMin, m_size, worldPos, n, scaling, &worldBB);
    m_worldBB = worldBB;
    m_worldToMedium = glm::inverse(m_mediumToWorld);
  }

  CNVDBMedium::CNVDBMedium(const std::string& path, const glm::vec3& sigma_a, const glm::vec3& sigma_s, float diffuseRoughness, float specularRoughness, const glm::vec3& worldPos, const glm::vec3& n, const glm::vec3& scaling) :
    CMedium(EMediumType::NVDB_MEDIUM),
    m_isHostObject(true),
    m_handle(getHandle(path)),
    m_grid(m_handle->grid<float>()),
    m_readAccessor(new nanovdb::DefaultReadAccessor<float>(m_grid->getAccessor())),
    m_deviceAabb(NULL),
    m_size(getMediumSize(m_grid->worldBBox(), m_grid->voxelSize())),
    m_sigma_a(sigma_a),
    m_sigma_s(sigma_s),
    m_phase(new CSGGXPhaseFunction(diffuseRoughness, specularRoughness)),
    m_sigma_t(sigma_a.z + sigma_s.z),
    m_invMaxDensity(1.f / getMaxValue(m_grid)),
    m_deviceResource(nullptr) {
    auto worldBBDim = m_grid->worldBBox().dim();
    auto voxelSize = m_grid->voxelSize();
    auto voxelCount = m_grid->activeVoxelCount();
    const nanovdb::CoordBBox box = m_grid->indexBBox();
    if (m_grid->activeVoxelCount() == 0) {
      m_ibbMin = glm::ivec3(0);
      m_ibbMax = glm::ivec3(0);
    }
    else {
      m_ibbMin = glm::ivec3(box.min().x(), box.min().y(), box.min().z());
      m_ibbMax = glm::ivec3(box.max().x(), box.max().y(), box.max().z());
    }
    nanovdb::BBoxR worldBB = m_grid->worldBBox();
    m_mediumToWorld = getMediumToWorldTransformation(m_grid->map(), m_ibbMin, m_size, worldPos, n, scaling, &worldBB);
    m_worldBB = worldBB;
    m_worldToMedium = glm::inverse(m_mediumToWorld);
  }

  CNVDBMedium::CNVDBMedium() :
    CMedium(EMediumType::NVDB_MEDIUM),
    m_isHostObject(true),
    m_handle(nullptr),
    m_grid(nullptr),
    m_readAccessor(nullptr),
    m_worldBB(),
    m_deviceAabb(NULL),
    m_size(0),
    m_mediumToWorld(1.f),
    m_worldToMedium(1.f),
    m_sigma_a(0.f),
    m_sigma_s(0.f),
    m_phase(nullptr),
    m_ibbMin(0),
    m_ibbMax(0),
    m_sigma_t(0.f),
    m_invMaxDensity(0.f),
    m_deviceResource(nullptr) {

  }

  CNVDBMedium::CNVDBMedium(CNVDBMedium&& medium) :
    CMedium(std::move(medium.type())),
    m_isHostObject(std::move(medium.m_isHostObject)),
    m_handle(std::exchange(medium.m_handle, nullptr)),
    m_grid(std::exchange(medium.m_grid, nullptr)),
    m_readAccessor(std::exchange(medium.m_readAccessor, nullptr)),
    m_worldBB(std::move(medium.m_worldBB)),
    m_deviceAabb(std::exchange(medium.m_deviceAabb, NULL)),
    m_size(std::move(medium.m_size)),
    m_mediumToWorld(std::move(medium.m_mediumToWorld)),
    m_worldToMedium(std::move(medium.m_worldToMedium)),
    m_sigma_a(std::move(medium.m_sigma_a)),
    m_sigma_s(std::move(medium.m_sigma_s)),
    m_phase(std::exchange(medium.m_phase, nullptr)),
    m_ibbMin(std::move(medium.m_ibbMin)),
    m_ibbMax(std::move(medium.m_ibbMax)),
    m_sigma_t(std::move(medium.m_sigma_t)),
    m_invMaxDensity(std::move(medium.m_invMaxDensity)),
    m_deviceResource(std::exchange(medium.m_deviceResource, nullptr)) {
  }

  CNVDBMedium::~CNVDBMedium() {
    if (m_isHostObject) {
      delete m_readAccessor;
      delete m_handle;
      delete m_phase;
    }
  }


  CNVDBMedium& CNVDBMedium::operator=(const CNVDBMedium&& medium) {
    return *this;
  }

  

  void CNVDBMedium::allocateDeviceMemory() {
    if (m_deviceResource) {
      freeDeviceMemory();
      delete m_deviceResource;
    }

    m_deviceResource = new DeviceResource();
    CUDA_ASSERT(hipMalloc(&m_deviceResource->d_readAccessor, sizeof(nanovdb::DefaultReadAccessor<float>)));
    switch (m_phase->type()) {
    case EPhaseFunction::HENYEY_GREENSTEIN:
      CUDA_ASSERT(hipMalloc(&m_deviceResource->d_phase, sizeof(CHenyeyGreensteinPhaseFunction)));
      break;
    case EPhaseFunction::SGGX:
      CUDA_ASSERT(hipMalloc(&m_deviceResource->d_phase, sizeof(CSGGXPhaseFunction)));
      break;
    }
  }

  CNVDBMedium CNVDBMedium::copyToDevice() const {
    m_handle->deviceUpload();

    
    CNVDBMedium medium;
    medium.m_isHostObject = false;
    medium.m_handle = this->m_handle;
    medium.m_grid = m_handle->deviceGrid<float>();
    if (!medium.m_grid) {
      fprintf(stderr, "GridHandle does not contain a valid device grid");
    }
    if (m_deviceResource) {
      medium.m_readAccessor = m_deviceResource->d_readAccessor;
      CUDA_ASSERT(hipMemcpy(m_deviceResource->d_readAccessor, this->m_readAccessor, sizeof(nanovdb::DefaultReadAccessor<float>), hipMemcpyHostToDevice));

      medium.m_phase = m_deviceResource->d_phase;
      switch (m_phase->type()) {
      case EPhaseFunction::HENYEY_GREENSTEIN:
        CUDA_ASSERT(hipMemcpy(m_deviceResource->d_phase, this->m_phase, sizeof(CHenyeyGreensteinPhaseFunction), hipMemcpyHostToDevice));
        break;
      case EPhaseFunction::SGGX:
        CUDA_ASSERT(hipMemcpy(m_deviceResource->d_phase, this->m_phase, sizeof(CSGGXPhaseFunction), hipMemcpyHostToDevice));
        break;
      }
    }
    else {
      medium.m_readAccessor = nullptr;
      fprintf(stderr, "No device resource for CNVDBMedium");
    }
    medium.m_size = this->m_size;
    medium.m_mediumToWorld = this->m_mediumToWorld;
    medium.m_worldToMedium = this->m_worldToMedium;
    medium.m_sigma_a = this->m_sigma_a;
    medium.m_sigma_s = this->m_sigma_s;
    medium.m_sigma_t = this->m_sigma_t;
    medium.m_invMaxDensity = this->m_invMaxDensity;
    medium.m_deviceResource = nullptr;
    medium.m_ibbMin = m_ibbMin;
    medium.m_ibbMax = m_ibbMax;
    medium.m_worldBB = m_worldBB;
    
    return medium;
  }

  void CNVDBMedium::freeDeviceMemory() const {
    if (m_deviceResource) {
      CUDA_ASSERT(hipFree(m_deviceResource->d_readAccessor));
      CUDA_ASSERT(hipFree(m_deviceResource->d_phase));
    }
    CUDA_ASSERT(hipFree(reinterpret_cast<void*>(m_deviceAabb)));
  }

  glm::ivec3 CNVDBMedium::getMediumSize(const nanovdb::BBox<nanovdb::Vec3R>& boundingBox, const nanovdb::Vec3R& voxelSize) {
    nanovdb::Vec3R size = boundingBox.dim() / voxelSize;
    return glm::ivec3(size[0], size[1], size[2]);
  }

  float CNVDBMedium::getMaxValue(const nanovdb::NanoGrid<float>* grid) {
    float min = 0.f;
    float max = 0.f;
    grid->tree().extrema(min, max);
    return max;
  }

  glm::mat4 CNVDBMedium::getMediumToWorldTransformation(const nanovdb::Map& map, const glm::ivec3& ibbMin, const glm::ivec3& size, const glm::vec3& worldPos, const glm::vec3& n, const glm::vec3& scaling, nanovdb::BBoxR* bbox) {
    glm::mat4 nanoIndexToWorld(map.mMatF[0], map.mMatF[3], map.mMatF[6], 0.f,
                               map.mMatF[1], map.mMatF[4], map.mMatF[7], 0.f,
                               map.mMatF[2], map.mMatF[5], map.mMatF[8], 0.f,
                               map.mVecF[0], map.mVecF[1], map.mVecF[2], 1.f); // [IdxMin, IdxMax] to world space
    glm::mat4 indexToNano((float)size[0], 0.f, 0.f, 0.f,
                          0.f, (float)size[1], 0.f, 0.f,
                          0.f, 0.f, (float)size[2], 0.f,
                          ibbMin.x, ibbMin.y, ibbMin.z, 1.f); // [0, 1] to [IdxMin, IdxMax] (Nanovdb index space
    glm::mat4 transformations = glm::translate(glm::mat4(1.0f), worldPos) * getRotation(n) * glm::scale(scaling);
    glm::vec4 newWorldMin = transformations * glm::vec4(bbox->min()[0], bbox->min()[1], bbox->min()[2], 1.f);
    glm::vec4 newWorldMax = transformations * glm::vec4(bbox->max()[0], bbox->max()[1], bbox->max()[2], 1.f);
    *bbox = nanovdb::BBoxR(nanovdb::Vec3R{ std::min(newWorldMin.x, newWorldMax.x), std::min(newWorldMin.y, newWorldMax.y), std::min(newWorldMin.z, newWorldMax.z) }, nanovdb::Vec3R{ std::max(newWorldMin.x, newWorldMax.x), std::max(newWorldMin.y, newWorldMax.y), std::max(newWorldMin.z, newWorldMax.z) });
    return transformations * nanoIndexToWorld * indexToNano;
  }

  nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>* CNVDBMedium::getHandle(const std::string& path) {
    nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>* handle = nullptr;
    try {
      handle = new nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>(nanovdb::io::readGrid<nanovdb::CudaDeviceBuffer>(path));
    }
    catch (const std::exception& e) {
      fprintf(stderr, "Couldn't load nvdb file: %s", e.what());
      handle = nullptr;
    }
    return handle;
  }

  SBuildInputWrapper CNVDBMedium::getOptixBuildInput() {
    if (!m_deviceAabb) {
      OptixAabb aabb{ m_worldBB.min()[0], m_worldBB.min()[1], m_worldBB.min()[2], m_worldBB.max()[0], m_worldBB.max()[1], m_worldBB.max()[2] };
      CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&m_deviceAabb), sizeof(OptixAabb)));
      CUDA_ASSERT(hipMemcpy(reinterpret_cast<void*>(m_deviceAabb), &aabb, sizeof(OptixAabb), hipMemcpyHostToDevice));
    }

    SBuildInputWrapper wrapper;
    wrapper.flags.push_back(OPTIX_GEOMETRY_FLAG_NONE);

    wrapper.buildInput = {};
    wrapper.buildInput.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
    wrapper.buildInput.customPrimitiveArray.aabbBuffers = &m_deviceAabb;
    wrapper.buildInput.customPrimitiveArray.flags = wrapper.flags.data();
    wrapper.buildInput.customPrimitiveArray.numSbtRecords = 1;
    wrapper.buildInput.customPrimitiveArray.numPrimitives = 1;
    wrapper.buildInput.customPrimitiveArray.sbtIndexOffsetBuffer = 0;
    wrapper.buildInput.customPrimitiveArray.sbtIndexOffsetSizeInBytes = 0;
    wrapper.buildInput.customPrimitiveArray.primitiveIndexOffset = 0;

    return wrapper;
  }

  OptixProgramGroup CNVDBMedium::getOptixProgramGroup() const {
    return CRTBackend::instance()->programGroups().m_hitVolume;
  }
}