#include "hip/hip_runtime.h"
#include "mesh/mesh.hpp"
#include "utility/debugging.hpp"
#include <backend/rt_backend.hpp>
namespace rt {
  CMesh::CMesh(const std::vector<glm::vec3>& vbo, const std::vector<glm::uvec3>& ibo, const std::vector<glm::vec3>& normals, const glm::vec3& bbMin, const glm::vec3& bbMax):
    m_deviceObject(false),
    m_deviceResource(nullptr) {
    CUDA_LOG_ERROR_STATE();
    m_numVertices = vbo.size();
    size_t vboBytes = sizeof(glm::vec3) * vbo.size();
    m_vbo = static_cast<glm::vec3*>(malloc(vboBytes));
    m_numIndices = ibo.size();
    size_t iboBytes = sizeof(glm::uvec3) * ibo.size();
    m_ibo = static_cast<glm::uvec3*>(malloc(iboBytes));
    size_t normalsBytes = sizeof(glm::vec3) * normals.size();
    m_normals = static_cast<glm::vec3*>(malloc(normalsBytes));

    uint64_t vboAdress = reinterpret_cast<uint64_t>(m_vbo);
    uint64_t iboAdress = reinterpret_cast<uint64_t>(m_ibo);
    size_t maxIndex = 0;
    for (auto index : ibo) {
      size_t triangleMax = glm::max(index.x, glm::max(index.y, index.z));
      maxIndex = glm::max(triangleMax, maxIndex);
    }

    glm::vec3 lastVertex = vbo.data()[maxIndex];

    memcpy(m_vbo, vbo.data(), vboBytes);
    memcpy(m_ibo, ibo.data(), iboBytes);
    memcpy(m_normals, normals.data(), normalsBytes);
    m_aabb = { bbMin.x, bbMin.y, bbMin.z, bbMax.x, bbMax.y, bbMax.z };

    //std::vector<glm::vec3> vbo2;
    //vbo2.push_back(glm::vec3(10.f, 0.f, 0.f));
    //vbo2.push_back(glm::vec3(0.f, 10.f, 0.f));
    //vbo2.push_back(glm::vec3(0.f, 0.f, 10.f));
    //std::vector<glm::uvec3> ibo2;
    //ibo2.push_back(glm::uvec3(0, 1, 2));
    //m_numVertices = vbo2.size();
    //size_t vboBytes = sizeof(glm::vec3) * vbo2.size();
    //m_vbo = static_cast<glm::vec3*>(malloc(vboBytes));
    //m_numIndices = ibo2.size();
    //size_t iboBytes = sizeof(glm::uvec3) * ibo2.size();
    //m_ibo = static_cast<glm::uvec3*>(malloc(iboBytes));
    //size_t normalsBytes = sizeof(glm::vec3) * normals.size();
    //m_normals = static_cast<glm::vec3*>(malloc(normalsBytes));

    //uint64_t vboAdress = reinterpret_cast<uint64_t>(m_vbo);
    //uint64_t iboAdress = reinterpret_cast<uint64_t>(m_ibo);
    //size_t maxIndex = 0;
    //for (auto index : ibo2) {
    //  size_t triangleMax = glm::max(index.x, glm::max(index.y, index.z));
    //  maxIndex = glm::max(triangleMax, maxIndex);
    //}

    //glm::vec3 lastVertex = vbo2.data()[maxIndex];

    //memcpy(m_vbo, vbo2.data(), vboBytes);
    //memcpy(m_ibo, ibo2.data(), iboBytes);
    //memcpy(m_normals, normals.data(), normalsBytes);
    //m_aabb = { bbMin.x, bbMin.y, bbMin.z, bbMax.x, bbMax.y, bbMax.z };
    CUDA_LOG_ERROR_STATE();
  }

  CMesh::CMesh():
    m_numVertices(0),
    m_vbo(nullptr),
    m_numIndices(0),
    m_ibo(nullptr),
    m_normals(nullptr),
    m_aabb{ 0.f, 0.f, 0.f, 0.f, 0.f, 0.f },
    m_deviceAabb(nullptr),
    m_deviceObject(false),
    m_deviceResource(nullptr) {

  }

  CMesh::CMesh(CMesh&& mesh):
    m_numVertices(std::move(mesh.m_numVertices)),
    m_vbo(std::exchange(mesh.m_vbo, nullptr)),
    m_numIndices(std::move(mesh.m_numIndices)),
    m_ibo(std::exchange(mesh.m_ibo, nullptr)),
    m_normals(std::exchange(mesh.m_normals, nullptr)),
    m_aabb(std::move(mesh.m_aabb)),
    m_deviceAabb(std::move(mesh.m_deviceAabb)),
    m_deviceObject(std::move(mesh.m_deviceObject)),
    m_deviceResource(std::move(mesh.m_deviceResource)) {

  }

  CMesh::~CMesh() {
    if (!m_deviceObject) {
      free(m_vbo);
      m_vbo = nullptr;
      free(m_ibo);
      m_ibo = nullptr;
      free(m_normals);
      m_normals = nullptr;
    }
  }

  void CMesh::allocateDeviceMemory() {
    if (m_deviceResource) {
      freeDeviceMemory();
      delete m_deviceResource;
    }
    m_deviceResource = new SMeshDeviceResource();
    hipMalloc(&m_deviceResource->d_vbo, sizeof(glm::vec3) * m_numVertices + 128);
    hipMalloc(&m_deviceResource->d_ibo, sizeof(glm::uvec3) * m_numIndices + 128);
    hipMalloc(&m_deviceResource->d_normals, sizeof(glm::vec3) * m_numVertices);
    CUDA_LOG_ERROR_STATE();
  }

  CMesh CMesh::copyToDevice() {
    CUDA_ASSERT(hipMemcpy(m_deviceResource->d_vbo, m_vbo, sizeof(glm::vec3) * m_numVertices, hipMemcpyHostToDevice));
    CUDA_ASSERT(hipMemcpy(m_deviceResource->d_ibo, m_ibo, sizeof(glm::uvec3) * m_numIndices, hipMemcpyHostToDevice));
    CUDA_ASSERT(hipMemcpy(m_deviceResource->d_normals, m_normals, sizeof(glm::vec3) * m_numVertices, hipMemcpyHostToDevice));

    CMesh deviceMesh;
    deviceMesh.m_numVertices = m_numVertices;
    deviceMesh.m_vbo = m_deviceResource->d_vbo;
    deviceMesh.m_numIndices = m_numIndices;
    deviceMesh.m_ibo = m_deviceResource->d_ibo;
    deviceMesh.m_normals = m_deviceResource->d_normals;
    deviceMesh.m_aabb = m_aabb;
    deviceMesh.m_deviceObject = true;
    return deviceMesh;
  }

  void CMesh::freeDeviceMemory() {
    if (m_deviceResource) {
      hipFree(m_deviceResource->d_vbo);
      hipFree(m_deviceResource->d_ibo);
      hipFree(m_deviceResource->d_normals);
    }
  }

  SBuildInputWrapper CMesh::getOptixBuildInput() {
    //if (!m_deviceAabb) {
      //CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&m_deviceAabb), sizeof(OptixAabb)));
      //CUDA_ASSERT(hipMemcpy(reinterpret_cast<void*>(m_deviceAabb), &m_aabb, sizeof(OptixAabb), hipMemcpyHostToDevice));
    //}
    if (m_deviceResource) {
      CUDA_ASSERT(hipMemcpy(reinterpret_cast<void*>(m_deviceResource->d_vbo), m_vbo, sizeof(glm::vec3) * m_numVertices, hipMemcpyHostToDevice)); // TODO: Find way to copy vertices and indices only once
      CUDA_ASSERT(hipMemcpy(reinterpret_cast<void*>(m_deviceResource->d_ibo), m_ibo, sizeof(glm::uvec3) * m_numIndices, hipMemcpyHostToDevice));
    }
    else {
      fprintf(stderr, "[ERROR] CMesh::getOptixBuildInput: vertex buffer and index buffer not allocated on device.\n");
    }
    CUDA_LOG_ERROR_STATE();
    m_deviceVertices = reinterpret_cast<hipDeviceptr_t>(m_deviceResource->d_vbo);
    m_deviceIndices = reinterpret_cast<hipDeviceptr_t>(m_deviceResource->d_ibo);
    SBuildInputWrapper wrapper;
    wrapper.flags.push_back(OPTIX_GEOMETRY_FLAG_NONE);

    wrapper.buildInput = {};
    wrapper.buildInput.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
    wrapper.buildInput.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
    wrapper.buildInput.triangleArray.vertexStrideInBytes = sizeof(glm::vec3);
    wrapper.buildInput.triangleArray.numVertices = m_numVertices;
    wrapper.buildInput.triangleArray.vertexBuffers = reinterpret_cast<hipDeviceptr_t*>(&m_deviceResource->d_vbo);
    wrapper.buildInput.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
    wrapper.buildInput.triangleArray.numIndexTriplets = m_numIndices;
    wrapper.buildInput.triangleArray.indexBuffer = reinterpret_cast<hipDeviceptr_t>(m_deviceResource->d_ibo);
    //hipDeviceptr_t iboPtr = reinterpret_cast<hipDeviceptr_t>(m_deviceResource->d_ibo);
    wrapper.buildInput.triangleArray.indexStrideInBytes = sizeof(glm::uvec3);
    wrapper.buildInput.triangleArray.numSbtRecords = 1;
    wrapper.buildInput.triangleArray.sbtIndexOffsetBuffer = 0;
    wrapper.buildInput.triangleArray.sbtIndexOffsetSizeInBytes = 0;
    wrapper.buildInput.triangleArray.primitiveIndexOffset = 0;
    wrapper.buildInput.triangleArray.flags = wrapper.flags.data();
    return wrapper;

  }

  OptixProgramGroup CMesh::getOptixProgramGroup() const {
    return CRTBackend::instance()->programGroups().m_hitMesh;
  }
}