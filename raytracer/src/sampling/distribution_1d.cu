#include  "sampling/distribution_1d.hpp"
#include "sampling/sampler.hpp"
#include "utility/functions.hpp"

namespace rt {
  CDistribution1D::CDistribution1D():
    m_func(nullptr),
    m_cdf(nullptr),
    m_deviceResource(nullptr) {

  }

  CDistribution1D::CDistribution1D(std::vector<float>& f):
    m_func(nullptr),
    m_cdf(nullptr),
    m_deviceResource(nullptr) {
    m_nFunc = f.size();
    m_func = new float[f.size()];
    memcpy(m_func, f.data(), f.size() * sizeof(float));

    m_nCdf = f.size() + 1;
    m_cdf = new float[m_nCdf];

    m_cdf[0] = 0;
    for (size_t i = 1; i < m_nCdf; ++i) {
      m_cdf[i] = m_cdf[i - 1] + m_func[i - 1] / m_nFunc;
    }

    m_funcInt = m_cdf[m_nFunc];
    if (m_funcInt == 0) {
      for (size_t i = 1; i < m_nCdf; ++i) {
        m_cdf[i] = (float)i / m_nFunc;
      }
    }
    else {
      for (size_t i = 1; i < m_nCdf; ++i) {
        m_cdf[i] /= m_funcInt;
      }
    }
  }

  CDistribution1D::~CDistribution1D() {
#ifndef __CUDA_ARCH__
    if (m_func) {
      delete m_func;
    }
    if (m_cdf) {
      delete m_cdf;
    }
#endif
    if (m_deviceResource) {
      freeDeviceMemory();
    }
  }

  float CDistribution1D::sampleContinuous(CSampler& sampler, float* pdf, size_t* off) const {
    float u = sampler.uniformSample01();

    auto& predicate = [&] (int index) { return m_cdf[index] <= u; };
    int offset = findInterval(m_nCdf, predicate);
    
    if (off) {
      *off = offset;
    }

    float du = u - m_cdf[offset];
    if ((m_cdf[offset + 1] - m_cdf[offset]) > 0) {
      du /= (m_cdf[offset + 1] - m_cdf[offset]);
    }

    if (pdf) {
      *pdf = m_func[offset] / m_funcInt;
    }

    return (offset + du) / count();
  }

  size_t CDistribution1D::sampleDiscrete(CSampler& sampler, float* pdf, float* uRemapped) const {
    float u = sampler.uniformSample01();

    auto& predicate = [&] (int index) { return m_cdf[index] <= u; };
    int offset = findInterval(m_nCdf, predicate);

    if (pdf) {
      *pdf = m_func[offset] / (m_funcInt * count());
    }

    if (uRemapped) {
      *uRemapped = (u - m_cdf[offset]) / (m_cdf[offset + 1] - m_cdf[offset]);
    }

    return offset;
  }

  float CDistribution1D::discretePdf(size_t index) const {
    float* func = m_func;
    float* cdf = m_cdf;
    return m_func[index] / (m_funcInt * count());
  }

  void CDistribution1D::copyToDevice(CDistribution1D* dst) {
    if (!m_deviceResource) {
      m_deviceResource = new SDistribution1D_DeviceResource();
      hipMalloc(&(m_deviceResource->d_func), sizeof(float) * m_nFunc);
      hipMalloc(&(m_deviceResource->d_cdf), sizeof(float) * m_nCdf);
    }

    hipMemcpy(m_deviceResource->d_func, m_func, sizeof(float) * m_nFunc, hipMemcpyHostToDevice);
    hipMemcpy(m_deviceResource->d_cdf, m_cdf, sizeof(float) * m_nCdf, hipMemcpyHostToDevice);

    CDistribution1D temp;
    temp.m_nFunc = m_nFunc;
    temp.m_func = m_deviceResource->d_func;
    temp.m_nCdf = m_nCdf;
    temp.m_cdf = m_deviceResource->d_cdf;
    temp.m_funcInt = m_funcInt;
    temp.m_deviceResource = nullptr;

    hipMemcpy(dst, &temp, sizeof(CDistribution1D), hipMemcpyHostToDevice);

    temp.m_func = nullptr;
    temp.m_cdf = nullptr;
  }

  void CDistribution1D::freeDeviceMemory() {
    if (m_deviceResource) {
      hipFree(m_deviceResource->d_func);
      hipFree(m_deviceResource->d_cdf);
      delete m_deviceResource;
      m_deviceResource = nullptr;
    }
  }


}