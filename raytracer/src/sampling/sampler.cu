#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include "..\..\include\sampling\sampler.hpp"

namespace rt {
  void CSampler::init() {
    hiprand_init(0, 0, 0, &m_curandState);
  }

  float CSampler::uniformSample01() {
    return hiprand_uniform(&m_curandState);
  }

  glm::vec3 CSampler::uniformSampleHemisphere() {
    float rand1 = hiprand_uniform(&m_curandState);
    float rand2 = hiprand_uniform(&m_curandState);

    float r = glm::sqrt(glm::max(0.0f, 1.0f - rand1 * rand1));
    float phi = 2.0 * M_PI * rand2;
    return glm::vec3(r * glm::cos(phi), r * glm::sin(phi), rand1);
  }

  float CSampler::uniformHemispherePdf() const {
    return 1.0f / (2 * M_PI);
  }

}
