#define _USE_MATH_DEFINES
#include <cmath>
#include "sampling/sampler.hpp"

namespace rt {
  void CSampler::init() {
    hiprand_init(0, 0, 0, &m_curandState);
  }

  void CSampler::init(uint64_t seed, uint64_t sequence) {
    hiprand_init(seed, sequence, 0, &m_curandState);
  }


}
