#include "scene/sceneobject.hpp"
#include <iostream>

#include "shapes/circle.hpp"
#include "shapes/sphere.hpp"
#include "medium/homogeneous_medium.hpp"
#include "shapes/rectangle.hpp"
#include "shapes/cuboid.hpp"
#include "medium/heterogenous_medium.hpp"

namespace rt {
  std::shared_ptr<CShape> CHostSceneobject::getShape(EShape shape, const glm::vec3& worldPos, float radius, const glm::vec3& normal) {
    switch (shape) {
    case EShape::CIRCLE:
      return std::make_shared<CCircle>(worldPos, radius, normal);
      break;
    case EShape::SPHERE:
      return std::make_shared<Sphere>(worldPos, radius, normal);
    }
  }

  CHostSceneobject::CHostSceneobject(const CShape* shape, const glm::vec3& le):
    m_shape(shape),
    m_material(nullptr),
    m_medium(nullptr),
    m_flag(ESceneobjectFlag::GEOMETRY),
    m_hostDeviceConnection(this) {
    m_material = std::make_shared<CMaterial>(CMaterial(le));
  }

  CHostSceneobject::CHostSceneobject(const CShape* shape, const glm::vec3& diffuseReflection, float diffuseRougness, const glm::vec3& specularReflection, float alphaX, float alphaY, float etaI, float etaT) :
    m_shape(shape),
    m_material(nullptr),
    m_medium(nullptr),
    m_flag(ESceneobjectFlag::GEOMETRY),
    m_hostDeviceConnection(this) {
    m_material = std::make_shared<CMaterial>(CMaterial(COrenNayarBRDF(diffuseReflection, diffuseRougness), CMicrofacetBRDF(specularReflection, alphaX, alphaY, etaI, etaT)));
  }

  CHostSceneobject::CHostSceneobject(const CShape* shape, CMedium* medium):
    m_shape(shape),
    m_material(nullptr),
    m_medium(medium),
    m_flag(ESceneobjectFlag::VOLUME),
    m_hostDeviceConnection(this) {
  }

  CHostSceneobject::CHostSceneobject(CHostSceneobject&& sceneobject) :
    m_shape(std::move(sceneobject.m_shape)),
    m_material(std::move(sceneobject.m_material)),
    m_medium(std::move(sceneobject.m_medium)),
    m_flag(std::move(sceneobject.m_flag)),
    m_hostDeviceConnection(this) {
  }

  CSceneobjectConnection::CSceneobjectConnection(CHostSceneobject* hostSceneobject):
    m_hostSceneobject(hostSceneobject) {
  }

  CSceneobjectConnection::CSceneobjectConnection(const CSceneobjectConnection&& connection) :
    m_hostSceneobject(std::move(connection.m_hostSceneobject)) {
  }
  void CSceneobjectConnection::allocateDeviceMemory() {
    switch (m_hostSceneobject->m_shape->shape()) {
    case EShape::CIRCLE:
      hipMalloc(&m_deviceShape, sizeof(CCircle));
      break;
    case EShape::SPHERE:
      hipMalloc(&m_deviceShape, sizeof(Sphere));
      break;
    case EShape::RECTANGLE:
      hipMalloc(&m_deviceShape, sizeof(CRectangle));
      break;
    case EShape::CUBOID:
      hipMalloc(&m_deviceShape, sizeof(CCuboid));
      break;
    }
    if (m_hostSceneobject->m_material) {
      hipMalloc(&m_deviceMaterial, sizeof(CMaterial));
    }
    if (m_hostSceneobject->m_medium) {
      switch (m_hostSceneobject->m_medium->type()) {
      case EMediumType::HOMOGENEOUS_MEDIUM:
        hipMalloc(&m_deviceMedium, sizeof(CHomogeneousMedium));
        break;
      case EMediumType::HETEROGENOUS_MEDIUM:
        hipMalloc(&m_deviceMedium, sizeof(CHeterogenousMedium));
        std::static_pointer_cast<CHeterogenousMedium>(m_hostSceneobject->m_medium)->allocateDeviceMemory();
        break;
      }
    }

  }
  void CSceneobjectConnection::copyToDevice() {
    switch (m_hostSceneobject->m_shape->shape()) {
    case EShape::CIRCLE:
      hipMemcpy(m_deviceShape, m_hostSceneobject->m_shape.get(), sizeof(CCircle), hipMemcpyHostToDevice);
      break;
    case EShape::SPHERE:
      hipMemcpy(m_deviceShape, m_hostSceneobject->m_shape.get(), sizeof(Sphere), hipMemcpyHostToDevice);
      break;
    case EShape::RECTANGLE:
      hipMemcpy(m_deviceShape, m_hostSceneobject->m_shape.get(), sizeof(CRectangle), hipMemcpyHostToDevice);
      break;
    case EShape::CUBOID:
      hipMemcpy(m_deviceShape, m_hostSceneobject->m_shape.get(), sizeof(CCuboid), hipMemcpyHostToDevice);
    }
    if (m_deviceMaterial) {
      hipMemcpy(m_deviceMaterial, m_hostSceneobject->m_material.get(), sizeof(CMaterial), hipMemcpyHostToDevice);
    }
    if (m_deviceMedium) {
      switch (m_hostSceneobject->m_medium->type()) {
      case EMediumType::HOMOGENEOUS_MEDIUM:
        hipMemcpy(m_deviceMedium, m_hostSceneobject->m_medium.get(), sizeof(CHomogeneousMedium), hipMemcpyHostToDevice);
        break;
      case EMediumType::HETEROGENOUS_MEDIUM:
        std::shared_ptr<CHeterogenousMedium> hetMedium = std::static_pointer_cast<CHeterogenousMedium>(m_hostSceneobject->m_medium);
        hipMemcpy(m_deviceMedium, &hetMedium->copyToDevice(), sizeof(CHeterogenousMedium), hipMemcpyHostToDevice);
        break;
      }
    }
    if (m_deviceSceneobject) {

      CDeviceSceneobject deviceSceneobject;
      deviceSceneobject.m_shape = m_deviceShape;
      deviceSceneobject.m_material = m_deviceMaterial;
      deviceSceneobject.m_medium = m_deviceMedium;
      deviceSceneobject.m_flag = m_hostSceneobject->m_flag;
      hipMemcpy(m_deviceSceneobject, &deviceSceneobject, sizeof(CDeviceSceneobject), hipMemcpyHostToDevice);
    }
  }

  void CSceneobjectConnection::freeDeviceMemory() {
    hipFree(m_deviceShape);
    hipFree(m_deviceMaterial);
    hipFree(m_deviceMedium);
  }

  SInteraction CDeviceSceneobject::intersect(const CRay& ray) {
    SInteraction si;
    switch (m_shape->shape()) {
    case EShape::CIRCLE:
      si.hitInformation = ((CCircle*)m_shape)->intersect(ray);
      break;
    case EShape::SPHERE:
      si.hitInformation = ((Sphere*)m_shape)->intersect(ray);
      break;
    case EShape::RECTANGLE:
      si.hitInformation = ((CRectangle*)m_shape)->intersect(ray);
      break;
    case EShape::CUBOID:
      si.hitInformation = ((CCuboid*)m_shape)->intersect(ray);
    }
    si.material = m_material;
    si.medium = m_medium;
    si.object = this;
    return si;
  }

  float CHostSceneobject::power() const {
    if (m_flag == ESceneobjectFlag::GEOMETRY) {
      glm::vec3 L = m_material->Le();
      switch (m_shape->shape()) {
      case EShape::CIRCLE:
        return (L.x + L.y + L.z) * ((CCircle*)m_shape.get())->area();
      }
    }
    return 0.0f;
  }

  CShape* CDeviceSceneobject::shape() const {
    return m_shape;
  }

  float CDeviceSceneobject::power() const {
    if (m_flag == ESceneobjectFlag::GEOMETRY) {
      glm::vec3 L = m_material->Le();
      switch (m_shape->shape()) {
      case EShape::CIRCLE:
        return (L.x + L.y + L.z) * ((CCircle*)m_shape)->area();
      }
    }
    return 0.0f;
  }
}