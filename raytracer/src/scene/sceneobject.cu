#include "scene/sceneobject.hpp"
#include <iostream>

#include "shapes/circle.hpp"
#include "shapes/sphere.hpp"
#include "medium/homogeneous_medium.hpp"
#include "shapes/rectangle.hpp"
#include "shapes/cuboid.hpp"
#include "medium/heterogenous_medium.hpp"
#include "medium/nvdb_medium.hpp"
#include "backend/rt_backend.hpp"
#include "utility/debugging.hpp"
#include <optix/optix_stubs.h>

namespace rt {
  std::shared_ptr<CShape> CHostSceneobject::getShape(EShape shape, const glm::vec3& worldPos, float radius, const glm::vec3& normal) {
    switch (shape) {
    case EShape::CIRCLE:
      return std::make_shared<CCircle>(worldPos, radius, normal);
      break;
    case EShape::SPHERE:
      return std::make_shared<Sphere>(worldPos, radius, normal);
    }
  }

  CHostSceneobject::CHostSceneobject(CShape* shape, const glm::vec3& le):
    m_shape(shape),
    m_mesh(nullptr),
    m_material(nullptr),
    m_medium(nullptr),
    m_flag(ESceneobjectFlag::GEOMETRY),
    m_deviceGasBuffer(NULL),
    m_hostDeviceConnection(this) {
    m_material = std::make_shared<CMaterial>(CMaterial(le));
  }

  CHostSceneobject::CHostSceneobject(CShape* shape, const glm::vec3& diffuseReflection, float diffuseRougness, const glm::vec3& specularReflection, float alphaX, float alphaY, float etaI, float etaT) :
    m_shape(shape),
    m_mesh(nullptr),
    m_material(nullptr),
    m_medium(nullptr),
    m_flag(ESceneobjectFlag::GEOMETRY),
    m_deviceGasBuffer(NULL),
    m_hostDeviceConnection(this) {
    m_material = std::make_shared<CMaterial>(CMaterial(COrenNayarBRDF(diffuseReflection, diffuseRougness), CMicrofacetBRDF(specularReflection, alphaX, alphaY, etaI, etaT)));
  }

  CHostSceneobject::CHostSceneobject(CShape* shape, CMedium* medium):
    m_shape(shape),
    m_mesh(nullptr),
    m_material(nullptr),
    m_medium(medium),
    m_flag(ESceneobjectFlag::VOLUME),
    m_deviceGasBuffer(NULL),
    m_hostDeviceConnection(this) {
  }

  CHostSceneobject::CHostSceneobject(CNVDBMedium* medium) :
    m_shape(nullptr),
    m_mesh(nullptr),
    m_material(nullptr),
    m_medium(medium),
    m_flag(ESceneobjectFlag::VOLUME),
    m_deviceGasBuffer(NULL),
    m_hostDeviceConnection(this) {
  }

  CHostSceneobject::CHostSceneobject(CHostSceneobject&& sceneobject) :
    m_shape(std::move(sceneobject.m_shape)),
    m_mesh(std::move(sceneobject.m_mesh)),
    m_material(std::move(sceneobject.m_material)),
    m_medium(std::move(sceneobject.m_medium)),
    m_flag(std::move(sceneobject.m_flag)),
    m_deviceGasBuffer(std::exchange(sceneobject.m_deviceGasBuffer, NULL)),
    m_hostDeviceConnection(this) {
  }

  CHostSceneobject::CHostSceneobject(CMesh* mesh, const glm::vec3& diffuseReflection, float diffuseRougness, const glm::vec3& specularReflection, float alphaX, float alphaY, float etaI, float etaT) :
    m_shape(nullptr),
    m_mesh(mesh),
    m_material(nullptr),
    m_medium(nullptr),
    m_flag(ESceneobjectFlag::GEOMETRY),
    m_deviceGasBuffer(NULL),
    m_hostDeviceConnection(this) {
    CUDA_LOG_ERROR_STATE();
    m_material = std::make_shared<CMaterial>(CMaterial(COrenNayarBRDF(diffuseReflection, diffuseRougness), CMicrofacetBRDF(specularReflection, alphaX, alphaY, etaI, etaT)));
    CUDA_LOG_ERROR_STATE();
  }

  CSceneobjectConnection::CSceneobjectConnection(CHostSceneobject* hostSceneobject):
    m_hostSceneobject(hostSceneobject) {
  }

  CSceneobjectConnection::CSceneobjectConnection(const CSceneobjectConnection&& connection) :
    m_hostSceneobject(std::move(connection.m_hostSceneobject)) {
  }

  void CSceneobjectConnection::allocateDeviceMemory() {
    CUDA_LOG_ERROR_STATE();
    if (m_hostSceneobject->m_shape) {
      switch (m_hostSceneobject->m_shape->shape()) {
      case EShape::CIRCLE:
        hipMalloc(&m_deviceShape, sizeof(CCircle));
        break;
      case EShape::SPHERE:
        hipMalloc(&m_deviceShape, sizeof(Sphere));
        break;
      case EShape::RECTANGLE:
        hipMalloc(&m_deviceShape, sizeof(CRectangle));
        break;
      case EShape::CUBOID:
        hipMalloc(&m_deviceShape, sizeof(CCuboid));
        break;
      }
    }
    if (m_hostSceneobject->m_mesh) {
      CUDA_LOG_ERROR_STATE();
      hipMalloc(&m_deviceMesh, sizeof(CMesh));
      m_hostSceneobject->m_mesh->allocateDeviceMemory();
      CUDA_LOG_ERROR_STATE();
    }
    if (m_hostSceneobject->m_material) {
      hipMalloc(&m_deviceMaterial, sizeof(CMaterial));
    }
    if (m_hostSceneobject->m_medium) {
      switch (m_hostSceneobject->m_medium->type()) {
      case EMediumType::HOMOGENEOUS_MEDIUM:
        hipMalloc(&m_deviceMedium, sizeof(CHomogeneousMedium));
        break;
      case EMediumType::HETEROGENOUS_MEDIUM:
        hipMalloc(&m_deviceMedium, sizeof(CHeterogenousMedium));
        std::static_pointer_cast<CHeterogenousMedium>(m_hostSceneobject->m_medium)->allocateDeviceMemory();
        break;
      case EMediumType::NVDB_MEDIUM:
        hipMalloc(&m_deviceMedium, sizeof(CNVDBMedium));
        std::static_pointer_cast<CNVDBMedium>(m_hostSceneobject->m_medium)->allocateDeviceMemory();
        break;
      }
    }
    
    CUDA_LOG_ERROR_STATE();
  }
  void CSceneobjectConnection::copyToDevice() {
    if (m_deviceShape) {
      switch (m_hostSceneobject->m_shape->shape()) {
      case EShape::CIRCLE:
        hipMemcpy(m_deviceShape, m_hostSceneobject->m_shape.get(), sizeof(CCircle), hipMemcpyHostToDevice);
        break;
      case EShape::SPHERE:
        hipMemcpy(m_deviceShape, m_hostSceneobject->m_shape.get(), sizeof(Sphere), hipMemcpyHostToDevice);
        break;
      case EShape::RECTANGLE:
        hipMemcpy(m_deviceShape, m_hostSceneobject->m_shape.get(), sizeof(CRectangle), hipMemcpyHostToDevice);
        break;
      case EShape::CUBOID:
        hipMemcpy(m_deviceShape, m_hostSceneobject->m_shape.get(), sizeof(CCuboid), hipMemcpyHostToDevice);
        break;
      }
    }
    if (m_deviceMesh) {
      hipMemcpy(m_deviceMesh, &m_hostSceneobject->m_mesh->copyToDevice(), sizeof(CMesh), hipMemcpyHostToDevice);
    }
    if (m_deviceMaterial) {
      hipMemcpy(m_deviceMaterial, m_hostSceneobject->m_material.get(), sizeof(CMaterial), hipMemcpyHostToDevice);
    }
    if (m_deviceMedium) {
      switch (m_hostSceneobject->m_medium->type()) {
      case EMediumType::HOMOGENEOUS_MEDIUM:
        hipMemcpy(m_deviceMedium, m_hostSceneobject->m_medium.get(), sizeof(CHomogeneousMedium), hipMemcpyHostToDevice);
        break;
      case EMediumType::HETEROGENOUS_MEDIUM: {
        std::shared_ptr<CHeterogenousMedium> hetMedium = std::static_pointer_cast<CHeterogenousMedium>(m_hostSceneobject->m_medium);
        hipMemcpy(m_deviceMedium, &hetMedium->copyToDevice(), sizeof(CHeterogenousMedium), hipMemcpyHostToDevice);
        break;
      }
      case EMediumType::NVDB_MEDIUM: {
        std::shared_ptr<CNVDBMedium> nvdbMedium = std::static_pointer_cast<CNVDBMedium>(m_hostSceneobject->m_medium);
        hipMemcpy(m_deviceMedium, &nvdbMedium->copyToDevice(), sizeof(CNVDBMedium), hipMemcpyHostToDevice);
        break;
      }
      }
    }
    if (m_deviceSceneobject) {

      CDeviceSceneobject deviceSceneobject;
      deviceSceneobject.m_shape = m_deviceShape;
      deviceSceneobject.m_mesh = m_deviceMesh;
      deviceSceneobject.m_material = m_deviceMaterial;
      deviceSceneobject.m_medium = m_deviceMedium;
      deviceSceneobject.m_flag = m_hostSceneobject->m_flag;
      hipMemcpy(m_deviceSceneobject, &deviceSceneobject, sizeof(CDeviceSceneobject), hipMemcpyHostToDevice);
    }
  }

  void CSceneobjectConnection::freeDeviceMemory() {
    hipFree(m_deviceShape);
    if (m_deviceMesh) {
      m_hostSceneobject->m_mesh->freeDeviceMemory();
      hipFree(m_deviceMesh);
    }
    hipFree(m_deviceMaterial);
    if (m_deviceMedium) {
      switch (m_hostSceneobject->m_medium->type()) {
        case EMediumType::HETEROGENOUS_MEDIUM: {
          std::shared_ptr<CHeterogenousMedium> hetMedium = std::static_pointer_cast<CHeterogenousMedium>(m_hostSceneobject->m_medium);
          hetMedium->freeDeviceMemory();
          break;
        }
        case EMediumType::NVDB_MEDIUM: {
          std::shared_ptr<CNVDBMedium> nvdbMedium = std::static_pointer_cast<CNVDBMedium>(m_hostSceneobject->m_medium);
          nvdbMedium->freeDeviceMemory();
          break;
        }
      }
      hipFree(m_deviceMedium);
    }
  }

  float CHostSceneobject::power() const {
    if (m_flag == ESceneobjectFlag::GEOMETRY) {
      glm::vec3 L = m_material->Le();
      switch (m_shape->shape()) {
      case EShape::CIRCLE:
        return (L.x + L.y + L.z) * ((CCircle*)m_shape.get())->area();
      }
    }
    return 0.0f;
  }

  CHostSceneobject::~CHostSceneobject() {
    CUDA_LOG_ERROR_STATE();
    hipFree((void*)m_deviceGasBuffer);
    CUDA_LOG_ERROR_STATE();
  }

  void CHostSceneobject::buildOptixAccel() {
    SBuildInputWrapper buildInputWrapper;
    if (m_medium.get() && m_medium->type() == NVDB_MEDIUM) {
      buildInputWrapper = ((CNVDBMedium*)m_medium.get())->getOptixBuildInput();
    }
    else if (m_mesh) {
      buildInputWrapper = m_mesh->getOptixBuildInput();
    }
    else if (m_shape) {
      buildInputWrapper = m_shape->getOptixBuildInput();
    }
    else {
      fprintf(stderr, "[ERROR] Could not create build input.\n");
    }

    OptixAccelBuildOptions accelOptions = {};
    accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
    accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

    OptixAccelBufferSizes gasBufferSizes;
    const OptixDeviceContext& context = CRTBackend::instance()->context();
    OPTIX_ASSERT(optixAccelComputeMemoryUsage(context, &accelOptions, &buildInputWrapper.buildInput, 1, &gasBufferSizes));
    CUDA_LOG_ERROR_STATE();

    hipDeviceptr_t d_tempBufferGas;
    CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&d_tempBufferGas), gasBufferSizes.tempSizeInBytes));
    hipDeviceptr_t d_outputBufferGas;
    CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&d_outputBufferGas), gasBufferSizes.outputSizeInBytes));
    hipDeviceptr_t d_compactedSize;
    CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&d_compactedSize), sizeof(size_t)));

    OptixAccelEmitDesc emitProperty = {};
    emitProperty.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
    emitProperty.result = d_compactedSize;

    printf("CUDA error state: %s\n", hipGetErrorString(hipGetLastError()));
    CUDA_LOG_ERROR_STATE();

    OPTIX_ASSERT(optixAccelBuild(CRTBackend::instance()->context(),
                                 0,
                                 &accelOptions,
                                 &buildInputWrapper.buildInput,
                                 1,
                                 d_tempBufferGas,
                                 gasBufferSizes.tempSizeInBytes,
                                 d_outputBufferGas,
                                 gasBufferSizes.outputSizeInBytes,
                                 &m_traversableHandle,
                                 &emitProperty,
                                 1));

    CUDA_ASSERT(hipStreamSynchronize(0));

    size_t compactedSize;
    CUDA_ASSERT(hipMemcpy(&compactedSize, reinterpret_cast<void*>(emitProperty.result), sizeof(size_t), hipMemcpyDeviceToHost));
    CUDA_ASSERT(hipFree(reinterpret_cast<void*>(d_compactedSize)));
    if (compactedSize < gasBufferSizes.outputSizeInBytes)
    {
      CUDA_ASSERT(hipMalloc(reinterpret_cast<void**>(&m_deviceGasBuffer), compactedSize));
      OPTIX_ASSERT(optixAccelCompact(context, 0, m_traversableHandle, m_deviceGasBuffer, compactedSize, &m_traversableHandle));
      CUDA_ASSERT(hipFree(reinterpret_cast<void*>(d_outputBufferGas)));
    }
    else
    {
      m_deviceGasBuffer = d_outputBufferGas;
    }
    CUDA_ASSERT(hipFree(reinterpret_cast<void*>(d_tempBufferGas)));
  }

  OptixInstance CHostSceneobject::getOptixInstance(uint32_t instanceId, uint32_t sbtOffset) const {
    OptixInstance instance;

    instance.flags = OPTIX_INSTANCE_FLAG_NONE;
    instance.instanceId = instanceId;
    instance.sbtOffset = sbtOffset;
    instance.visibilityMask = 0xff; // TODO: Check what has to be set here
    instance.traversableHandle = m_traversableHandle;
    float identity[] = { 1.f, 0.f, 0.f, 0.f,
                         0.f, 1.f, 0.f, 0.f,
                         0.f, 0.f, 1.f, 0.f };
    memcpy(instance.transform, identity, sizeof(float) * 12);

    return instance;
  }

  OptixProgramGroup CHostSceneobject::getOptixProgramGroup() const {
    if (m_medium.get()) {
      return m_medium->getOptixProgramGroup();
    }
    else if (m_shape.get()) {
      return m_shape->getOptixProgramGroup();
    }
    else if (m_mesh.get()) {
      return m_mesh->getOptixProgramGroup();
    }
    fprintf(stderr, "[ERROR] CHostSceneobject::getOptixProgramGroup no valid program group found.\n");
    return OptixProgramGroup();
  }

  SRecord<const CDeviceSceneobject*> CHostSceneobject::getSBTHitRecord() const {
    SRecord<const CDeviceSceneobject*> hitRecord;
    OPTIX_ASSERT(optixSbtRecordPackHeader(getOptixProgramGroup(), &hitRecord));
    CUDA_LOG_ERROR_STATE();
    if (!m_hostDeviceConnection.deviceSceneobject()) {
      fprintf(stderr, "[ERROR] CHostSceneobject::getSBTHitRecord: deviceSceneobject is null.\n");
    }
    hitRecord.data = m_hostDeviceConnection.deviceSceneobject();
    return hitRecord;
  }


}