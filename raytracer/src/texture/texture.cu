#include "texture/texture.hpp"

#include <glm/gtc/type_ptr.hpp>

#define STB_IMAGE_IMPLEMENTATION
#include <stb/stb_image.h>

namespace rt {
  CTexture::CTexture(): m_width(0), m_height(0), m_channels(0), m_data(nullptr), m_deviceResource(nullptr) {

  }

  CTexture::CTexture(const std::string& path): m_deviceResource(nullptr) {
    m_data = stbi_loadf(path.c_str(), &m_width, &m_height, &m_channels, 0);
  }

  CTexture::~CTexture() {
#ifndef __CUDA_ARCH__
    //delete[] m_data; // TODO: clean up object
#endif
    if (m_deviceResource) {
      freeDeviceMemory();
      delete m_deviceResource;
    }
  }

  

  void CTexture::allocateDeviceMemory() {
    if (m_deviceResource) {
      freeDeviceMemory();
      delete m_deviceResource;
    }
    m_deviceResource = new STexture_DeviceResource;
    hipMalloc(&m_deviceResource->d_data, sizeof(float) * m_width * m_height * m_channels);
  }

  CTexture CTexture::copyToDevice() const {
    if (m_deviceResource) {
      hipMemcpy(m_deviceResource->d_data, m_data, sizeof(float) * m_width * m_height * m_channels, hipMemcpyHostToDevice);
    }

    CTexture t;
    t.m_width = m_width;
    t.m_height = m_height;
    t.m_channels = m_channels;
    t.m_data = m_deviceResource->d_data;
    return t;
  }

  void CTexture::freeDeviceMemory() const {
    if (m_deviceResource) {
      hipFree(m_deviceResource->d_data);
    }
  }
}