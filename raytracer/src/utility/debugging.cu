#include "hip/hip_runtime.h"
#include "utility/debugging.hpp"
#include "sampling/sampler.hpp"
#include <string>
#include <vector>
#include <fstream>

namespace rt {
  __global__ void copyStates(CSampler* samplers, hiprandState_t* states, size_t numStates) {
    for (size_t i = 0; i < numStates; ++i) {
      states[i] = samplers[i].m_curandState;
    }
  }

  __global__ void copyStates(hiprandState_t* states, CSampler* samplers, size_t numStates) {
    for (size_t i = 0; i < numStates; ++i) {
      samplers[i].m_curandState = states[i];
    }
  }

  void storeRandomState(CSampler* samplers, size_t numStates, const char* storePath) {
    hiprandState_t* d_states;
    hipMalloc(&d_states, sizeof(hiprandState_t) * numStates);

    copyStates << <1, 1 >> > (samplers, d_states, numStates);
    CUDA_ASSERT(hipDeviceSynchronize());
    std::vector<hiprandState_t> localStates(numStates);
    hipMemcpy(localStates.data(), d_states, sizeof(hiprandState_t) * numStates, hipMemcpyDeviceToHost);

    hipFree(d_states);

    std::fstream s(storePath, std::ios_base::binary | std::ios_base::out);
    s.write((const char*)localStates.data(), sizeof(hiprandState_t) * localStates.size());
  }

  void loadRandomState(CSampler* samplers, const char* loadPath) {
    std::ifstream file(loadPath, std::ios::binary | std::ios::ate);
    std::streamsize size = file.tellg();
    file.seekg(0, std::ios::beg);

    std::vector<hiprandState_t> buffer(size / sizeof(hiprandState_t));
    file.read((char*)buffer.data(), size);

    hiprandState_t* d_states;
    hipMalloc(&d_states, size);

    hipMemcpy(d_states, buffer.data(), size, hipMemcpyHostToDevice);

    copyStates << <1, 1 >> > (d_states, samplers, size / sizeof(hiprandState_t));
    CUDA_ASSERT(hipDeviceSynchronize());

    hipFree(d_states);
  }
}